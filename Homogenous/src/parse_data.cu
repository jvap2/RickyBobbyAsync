#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"

#define BLOCKS 16
#define TPB 256






__host__ void return_edge_list(string path, edge* arr){
    ifstream data;
    data.open(path);
    string line,word;
    unsigned int count=0;
    unsigned int column=0;
    cout<<data.is_open()<<endl;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); //Stream Class to operate on strings
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        arr[count-1].start=stoi(word);
                        column++;
                    }
                    else{
                        arr[count-1].end=stoi(word);
                    }
                }
                //Extract data until ',' is found
            }
            column=0;
            count++;
        }
    }
    else{
        cout<<"Cannot open file"<<endl;
    }
    data.close();
}


__host__ void split_list(unsigned int** arr, unsigned int* subarr_1, unsigned int* subarr_2, unsigned int size){
    for(unsigned int i=0; i<size;i++){
        subarr_1[i]=arr[i][0];
        subarr_2[i]=arr[i][1];
    }
}


__global__ void Sort_Cluster(edge* edgelist, unsigned int* table, unsigned int size,unsigned int iter){
    //Need to sort through the cluster data and organize it
    //organize into the data for each block of FrogWild
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    __shared__ edge shared_edge[TPB];
    __shared__ unsigned int bits[TPB];
    __shared__ unsigned int ex_bits[TPB];
    //Load vertex and cluster info into the shared memory
    if(idx<size){
        shared_edge[tid]=edgelist[idx];
    }
    __syncthreads();

    //Perform sorting
    unsigned int key, bit, vert_val;
    int from, to;
    if(idx<size){
        key=shared_edge[tid].cluster;
        from = shared_edge[tid].start;
        to = shared_edge[tid].end;
        bit=(key>>iter) & 1;
        bits[tid]=bit;
    }
    __syncthreads();
    //Perform exclusive scan
    if(tid<TPB && tid!=0){
        ex_bits[tid]=bits[tid-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    if(tid<TPB){
        bits[tid]=ex_bits[tid];
    }
    __syncthreads();
    if(idx<size){
        unsigned int num_one_bef=bits[tid];
        unsigned int num_one_total=bits[TPB-1];
        unsigned int dst = (bit==0)? (tid - num_one_bef):(TPB-num_one_total-num_one_bef);
        shared_edge[dst].cluster=key;
        shared_edge[dst].start=from;
        shared_edge[dst].end=to;
    }
    __syncthreads();
    if(tid==0){
        table[blockIdx.x]=blockDim.x-bits[blockDim.x-1];
        //Save the number of 1's
        table[blockIdx.x+gridDim.x]=bits[blockDim.x-1];
    }
    __syncthreads();
    if(idx==0){
        //Have thread 0 launch the kernel to perform the sum
        //Save the number of 0's
        bit_exclusive_scan<<<1,2*gridDim.x,0,cudaStreamTailLaunch>>>(table,2*gridDim.x);
    }
    __syncthreads();
    // // //We now have the pointer values in global memory to store data
    if(idx<size){
        if(tid<=blockDim.x-bits[blockDim.x-1]){
            edgelist[table[blockIdx.x]+tid]=shared_edge[tid];
        }
        else{
            edgelist[table[blockIdx.x+gridDim.x]+tid]=shared_edge[tid];
        }
    }
    __syncthreads();
}

__global__ void Swap(unsigned int* cluster, unsigned int* vertex, unsigned int* table, unsigned int* table_2, unsigned  int size){
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    // const unsigned int cluster_size= size/gridDim.x+1;
    __shared__ unsigned int shared_edge[TPB];
    __shared__ unsigned int shared_vertex[TPB];
    //Load vertex and cluster info into the shared memory
    if(idx<size){
        shared_edge[tid]=cluster[idx];
        shared_vertex[tid]=vertex[idx];
    }
    __syncthreads();   
    if(idx<size){
        if(tid<=table_2[blockIdx.x]){
            cluster[table[blockIdx.x]+tid]=shared_edge[tid];
            vertex[table[blockIdx.x]+tid]=shared_vertex[tid];
        }
        else{
            cluster[table[blockIdx.x+gridDim.x]+tid]=shared_edge[tid];
            vertex[table[blockIdx.x+gridDim.x]+tid]=shared_vertex[tid];
        }
    }
    __syncthreads();
}

__global__ void bit_exclusive_scan(unsigned int* bits,unsigned int size){
    unsigned int tid=threadIdx.x;
    __shared__ unsigned int ex_bits[TPB];
    if(tid<size && tid!=0){
        ex_bits[tid]=bits[tid-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    if(tid<TPB){
        // bit_2[tid]=ex_bits[tid];
        bits[tid]=ex_bits[tid];
    }
    __syncthreads();
}



__host__ void Org_Vertex_Helper(edge* h_edge, int size){
    //Allocate memory for vertex and cluster info
    edge* d_edge;
    unsigned int* d_table;
    // unsigned int* d_table_2;

    unsigned int threads_per_block=TPB;
    unsigned int blocks_per_grid= size/threads_per_block+1;

    if(!HandleCUDAError(hipMalloc((void**) &d_edge, size*sizeof(edge)))){
        cout<<"Unable to allocate memory for vertex data"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**) &d_table,(2*blocks_per_grid)*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table,0,(2*blocks_per_grid)*sizeof(unsigned int)))){
        cout<<"Unable to set table to 0"<<endl;
    }

    if(!HandleCUDAError(hipMemcpy(d_edge,h_edge,size*sizeof(edge), hipMemcpyHostToDevice))){
        cout<<"Unable to copy cluster data"<<endl;
    }
    double r = ((double) rand() / (RAND_MAX));
    Random_Edge_Placement<<<blocks_per_grid,threads_per_block>>>(d_edge, r);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Rand_Edge Place"<<endl;
    } 
    for(unsigned int i=0; i<32;i++){
        Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
        }
        // bit_exclusive_scan<<<1,2*blocks_per_grid>>>(d_table,d_table_2,2*blocks_per_grid);
        // if(!HandleCUDAError(hipDeviceSynchronize())){
        //     cout<<"Unable to synchronize with host exclusive scan"<<endl;
        // }
        // Swap<<<blocks_per_grid,threads_per_block>>>(d_cluster,d_vertex,d_table_2,d_table,size);
        // if(!HandleCUDAError(hipDeviceSynchronize())){
        //     cout<<"Unable to synchronize with host swap"<<endl;
        // }
    }

    if(!HandleCUDAError(hipMemcpy(h_edge,d_edge,size*sizeof(unsigned int),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back vertex data"<<endl;
    }
    HandleCUDAError(hipFree(d_edge));
    HandleCUDAError(hipFree(d_table));
    HandleCUDAError(hipDeviceReset());   
}



__host__ graph *create_graph (edge *edges){
   int i;
   struct graph *graph = (struct graph *) malloc (sizeof (struct graph));
   for (i = 0; i < NODES; i++) {
      graph->point[i] = NULL;
   }
   for (i = 0; i < EDGES; i++) {
      int start = edges[i].start;
      int end = edges[i].end;
      struct vertex *v = (struct vertex *) malloc (sizeof (struct vertex));
      v->end = end;
      v->next = graph->point[start];
      graph->point[start] = v;
   }
   return graph;
}

__global__ void Random_Edge_Placement(edge *edges, double rand_num){
    unsigned int idx= threadIdx.x+blockDim.x*blockIdx.x;
    __syncthreads();
    //Use multiplication hashing
    double intpart;
    double mod_part = modf(idx*rand_num, &intpart);
    unsigned int hash = (unsigned int)(BLOCKS*mod_part);
    //We now have the key, we need to sort
    if(idx<EDGES){
        edges[idx].cluster=hash;
    }
    __syncthreads();

}