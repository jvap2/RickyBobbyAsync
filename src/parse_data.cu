#include "hip/hip_runtime.h"
#include "../include/data.h"



__host__ void return_list(string path, int** arr){
    fstream data;
    data.open(path);
    string line,word;
    int count=0;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); //Stream Class to operate on strings
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    *(arr[count-1])=stoi(word);
                    arr[count-1]++;
                }
                //Extract data until ',' is found
            }
            count++;
        }
    }
    data.close();
}

__host__ void split_list(int** arr, int* subarr_1, int* subarr_2, int size){
    for(int i=0; i<size;i++){
        subarr_1[i]=arr[i][0];
        subarr_2[i]=arr[i][1];
    }
}


__global__ void Sort_Cluster(int* cluster, int* vertex, int* cluster_out, int* vertex_out, int* bits, int size, int iter){
    //Need to sort through the cluster data and organize it
    //organize into the data for each block of FrogWild
    int idx= threadIdx.x + blockIdx.x*blockDim.x;
    int tid= threadIdx.x;
    int cluster_size= size/gridDim.x+1;
    __shared__ int shared_cluster[cluster_size];
    __shared__ int shared_vertex[cluster_size];
    //Load vertex and cluster info into the shared memory
    if(idx<size){
        shared_cluster[tid]=cluster[idx];
        shared_vertex[tid]=vertex[idx];
    }
    __syncthreads();

    //Perform sorting
    unsigned int key, bit;
    if(idx<size){
        key=shared_cluster[tid];
        bit=(key>>iter) & 1;

    }


}


