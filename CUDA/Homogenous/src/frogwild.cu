#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"

#define TPB 128

__global__ void acc_accum(unsigned int* approx, unsigned int* pagerank, unsigned int* table, unsigned int k){
    unsigned int idx=threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid = threadIdx.x;
    __shared__ unsigned int local_table[TPB];
    if(idx<k){
        if(approx[idx]==pagerank[idx]){
            local_table[tid]=1;
        }
        else{
            local_table[tid]=0;
        }
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=local_table[tid]+local_table[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            local_table[tid]=temp;
        }
    }
    if(tid==(blockDim.x-1)){
        local_table[tid]=table[idx];
    }
}

__global__ float fin_acc(unsigned int* table, unsigned int k){
    float acc;
    unsigned int tid = threadIdx.x;
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=table[tid]+table[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            table[tid]=temp;
        }
    }
    if(tid==(blockDim.x-1)){
        acc=float(table[tid])/float(k);
    }
    return acc;
}

