#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"

#define thrd_blck 512

__host__ void Import_Local_Src(unsigned int* local_src){
    ifstream myfile;
    myfile.open(LOCAL_SRC_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        local_src[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}


__host__ void Import_Local_Succ(unsigned int* local_succ){
    ifstream myfile;
    myfile.open(LOCAL_SUCC_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        local_succ[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}

__host__ void Import_Unique(unsigned int* unq){
    ifstream myfile;
    myfile.open(UNQ_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        unq[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}



__host__ void Import_Src_Ctr_Ptr(unsigned int* src_ctr, unsigned int* src_ptr){
    ifstream myfile;
    myfile.open(SRC_CTR_PTR_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        src_ctr[count-1] = stoi(word);
                        column++;
                    }
                    else{
                        src_ptr[count-1] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }

}


__host__ void Import_Unq_Ptr_Ctr(unsigned int* unq_ptr, unsigned int* unq_ctr){
    ifstream myfile;
    myfile.open(UNQ_CTR_PTR_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        unq_ctr[count-1] = stoi(word);
                        column++;
                    }
                    else{
                        unq_ptr[count-1] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }

}


__host__ void Import_H_Ctr_Ptr(unsigned int* h_ctr, unsigned int* h_ptr){
    ifstream myfile;
    myfile.open(H_CTR_PTR_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        h_ctr[count-1] = stoi(word);
                        column++;
                    }
                    else{
                        h_ptr[count-1] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }

}


__host__ void Import_Degree(unsigned int* deg, unsigned int node_size){
    ifstream myfile;
    myfile.open(DEG_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        deg[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}

__host__ void Import_Replica_Stats(replica_tracker* h_replica, unsigned int node_size){
    ifstream myfile;
    myfile.open(REPLICA_STAT_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        h_replica[count-1].num_replicas = stoi(word);
                        column++;
                    }
                    else if(column>1 && column <BLOCKS+2){
                        h_replica[count-1].clusters[column-2] = stoi(word);
                    }
                    else{
                        h_replica[count-1].master_rep= stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }
}

__host__ void Import_Global_Src(unsigned int* src){
    ifstream myfile;
    myfile.open(GLOBAL_SRC_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    src[count-1] = stoi(word);
                }
            }
            column = 0;
            count++;
        }
    }
}


__host__ void Import_Global_Succ(unsigned int* succ){
    ifstream myfile;
    myfile.open(GLOBAL_SUCC_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    succ[count-1] = stoi(word);
                }
            }
            column = 0;
            count++;
        }
    }
}


__host__ void Export_C(unsigned int* c, unsigned int* indices, unsigned int node_size){
    ofstream myfile;
    myfile.open(C_PATH);
    myfile<<"Node,Count"<<endl;
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        for(unsigned int i=0; i<node_size; i++){
            myfile<<indices[i]<<","<<c[i]<<endl;
        }
    }
}

__host__ void Export_K(unsigned int* k, unsigned int node_size){
    ofstream myfile;
    myfile.open(K_PATH);
    if(!myfile.is_open()){
        std::cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        for(unsigned int i=0; i<node_size; i++){
            myfile<<k[i]<<endl;
        }
    }
}


__host__ void FrogWild(unsigned int* local_succ, unsigned int* local_src, unsigned int* unq, unsigned int* c, unsigned int* k, unsigned int* src_ptr, 
unsigned int* unq_ptr, unsigned int* h_ptr, unsigned int* degree, unsigned int* global_src, unsigned int* global_succ,
replica_tracker* h_replica, int node_size, unsigned int edge_size, unsigned int max_unq_ctr, unsigned int version,
unsigned int* ind_rank, unsigned int debug){
    int deviceCount=0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
                static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    int max_num_threads=0;
    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        max_num_threads=deviceProp.maxThreadsPerBlock;
    }
    unsigned int *d_succ, *d_src, *d_unq, *d_c, *d_k, *d_src_ptr, *d_unq_ptr, *d_h_ptr, *d_degree, *d_global_src, *d_global_succ;
    replica_tracker *d_replica;
    float p_t, p_s;
    p_s=.8;
    p_t=.8;
    unsigned int iter = 5;
    float* d_p_t, *d_p_s;
    unsigned int unq_ctr_max=0;
    unsigned int src_ctr_max=0;
    unsigned int h_ctr_max=0;
    unsigned int *num_local_K;
    unsigned int *num_local_C;
    for(int i = 1; i<=BLOCKS;i++){
        if(unq_ptr[i]-unq_ptr[i-1]>unq_ctr_max){
            unq_ctr_max=unq_ptr[i]-unq_ptr[i-1];
        }
        if(h_ptr[i]-h_ptr[i-1]>h_ctr_max){
            h_ctr_max=h_ptr[i]-h_ptr[i-1];
        }
        if(src_ptr[i]-src_ptr[i-1]>src_ctr_max){
            src_ctr_max=src_ptr[i]-src_ptr[i-1];
        }
    }
    unsigned int *local_K;
    unsigned int *local_C;
    unsigned int *local_K_idx;
    unsigned int *local_C_idx;
    unsigned int *mirror_ctr;
    std::cout<<"Allocating memory for device variables"<<endl;
    if(version==0){
        if(!HandleCUDAError(hipMalloc((void**)&d_unq, (unq_ptr[BLOCKS])*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_unq"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_c, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_c"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_k, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_k"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_unq_ptr, (BLOCKS+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_unq_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_replica, node_size*sizeof(replica_tracker)))){
            std::cout<<"Error allocating memory for d_replica"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_p_t, sizeof(float)))){
            std::cout<<"Error allocating memory for d_p_t"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_p_s, sizeof(float)))){
            std::cout<<"Error allocating memory for d_p_s"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_K, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_K"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_C, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_C"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_global_src, (node_size+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_global_src"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_global_succ, (edge_size)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_global_succ"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&mirror_ctr, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for the mirror ctr"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_unq, unq, (unq_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_unq"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_unq_ptr, unq_ptr, (BLOCKS+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_unq_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_replica, h_replica, node_size*sizeof(replica_tracker), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_replica"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_p_t, &p_t, sizeof(float), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_p_t"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_p_s, &p_s, sizeof(float), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_p_s"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_global_src, global_src, (node_size+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_global_src"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_global_succ, global_succ, (edge_size)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_global_succ"<<endl;
        }
        float* rand_frog;
        int sublinear_size=node_size/10+1;
        std::cout<<"Sublinear size "<<sublinear_size<<endl;
        std::cout<<"Node size "<<node_size<<endl;
        if(!HandleCUDAError(hipMalloc((void**)&rand_frog, sublinear_size*sizeof(float)))){
            std::cout<<"Error allocating memory for rand_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemset(rand_frog, 0, sublinear_size*sizeof(float)))){
            std::cout<<"Error initializing rand_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemset(d_k, 0, node_size*sizeof(unsigned int)))){
            std::cout<<"Error initializing d_k"<<endl;
        }
        if(!HandleCUDAError(hipMemset(d_c, 0, node_size*sizeof(unsigned int)))){
            std::cout<<"Error initializing d_c"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_K, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_K"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_C, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_C"<<endl;
        }
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, rand_frog, sublinear_size);
        /*Now, we have the random numbers generated*/
        hiprandDestroyGenerator(gen);
        unsigned int t_per_block = thrd_blck;
        unsigned int b_per_grid_int = (sublinear_size+thrd_blck-1)/thrd_blck;
        unsigned int b_per_grid = (node_size+thrd_blck-1)/thrd_blck;
        hiprandState* d_state_teleport;
        if(!HandleCUDAError(hipMalloc((void**)&d_state_teleport, BLOCKS*thrd_blck*sizeof(hiprandState)))){
            std::cout<<"Error allocating memory for d_state"<<endl;
        }
        hiprandState* d_state_scatter;
        if(!HandleCUDAError(hipMalloc((void**)&d_state_scatter, BLOCKS*thrd_blck*sizeof(hiprandState)))){
            std::cout<<"Error allocating memory for d_state"<<endl;
        }
        std::cout<<"First init device configuration parameters"<<endl;
        std::cout<<"No. Blocks "<<b_per_grid_int<<endl;
        std::cout<<"No. Threads per block "<<t_per_block<<endl;
        First_Init<<<b_per_grid_int, t_per_block>>>(rand_frog, d_k, node_size, sublinear_size);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            std::cout<<"Error synchronizing device"<<endl;
        }
        hipError_t err_0= hipGetLastError();
        if (err_0 != hipSuccess) 
            printf("First_Init Error: %s\n", hipGetErrorString(err_0));
        hipEvent_t start, stop;
        if(!HandleCUDAError(hipEventCreate(&start))){
            std::cout<<"Error creating start event"<<endl;
        }
        if(!HandleCUDAError(hipEventCreate(&stop))){
            std::cout<<"Error creating stop event"<<endl;
        }
        if(!HandleCUDAError(hipEventRecord(start))){
            std::cout<<"Error recording start event"<<endl;
        }
        unsigned int* d_k_local_temp;
        if(!HandleCUDAError(hipMalloc((void**)&d_k_local_temp, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_k_temp"<<endl;
        }
        size_t free_byte ;
        size_t total_byte ;
        if(!HandleCUDAError(hipMemGetInfo( &free_byte, &total_byte ))){
            std::cout<<"Error getting memory info"<<endl;
        }
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        printf("GPU memory usage before PR: used = %f, free = %f MB, total = %f MB\n",
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
        // hipFuncSetAttribute(reinterpret_cast<const void*>(Apply_Ver0), hipFuncAttributeMaxDynamicSharedMemorySize, 102400);
        std::cout<<"CUDA Dimensions"<<endl;
        std::cout<<"No. Blocks "<<BLOCKS<<endl;  
        std::cout<<"No. Threads per block "<<t_per_block<<endl;
        for(unsigned int i=0; i<iter; i++){
            std::cout<<"Iteration "<<i<<endl;
            Gather_Ver0<<<BLOCKS,thrd_blck>>>(d_k, d_unq, d_unq_ptr, local_K);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) 
                printf("Gather Error: %s\n", hipGetErrorString(err));
            if(!HandleCUDAError(hipDeviceSynchronize())){
                std::cout<<"Error synchronizing device"<<endl;
            }
            if(!HandleCUDAError(hipMemset(d_k,0, node_size*sizeof(unsigned int)))){
                std::cout<<"Error initializing d_k"<<endl;
            }
            if(!HandleCUDAError(hipMemset(d_k_local_temp, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
                std::cout<<"Error initializing d_k_temp"<<endl;
            }
            std::cout<<"Gathered"<<endl;
            std::cout<<max_unq_ctr*sizeof(unsigned int)<<endl;
            Apply_Ver0<<<BLOCKS, thrd_blck>>>(d_unq_ptr, local_K,d_k_local_temp, local_C, d_p_t,i, d_state_teleport);
            hipError_t err1 = hipGetLastError();
            if (err1 != hipSuccess) 
                printf("Apply Error: %s\n", hipGetErrorString(err1));
            if(!HandleCUDAError(hipDeviceSynchronize())){
                std::cout<<"Error synchronizing device for Apply"<<endl;
            }
            std::cout<<"Applied"<<endl;
            Sync_Mirrors_Ver0<<<BLOCKS,thrd_blck>>>(d_c, d_k, d_unq, d_unq_ptr, local_C, local_K, d_global_src,d_global_succ,mirror_ctr,
            d_replica,node_size, d_p_s, d_state_scatter);
            hipError_t err2 = hipGetLastError();  
            if (err2 != hipSuccess) 
                printf("Sync Error: %s\n", hipGetErrorString(err2));
            if(!HandleCUDAError(hipDeviceSynchronize())){
                std::cout<<"Error synchronizing device for Sync"<<endl;
            }
            std::cout<<"Synced"<<endl;
            // Scatter_Ver0<<<b_per_grid,thrd_blck>>>(d_c, d_k, d_global_src, d_global_succ, d_replica, node_size);
            // err2= hipGetLastError();
            // if (err2 != hipSuccess) 
            //     printf("Error: %s\n", hipGetErrorString(err2));
            // if(!HandleCUDAError(hipDeviceSynchronize())){
            //     std::cout<<"Error synchronizing device for Scatter"<<endl;
            // }
            std::cout<<"Scattered"<<endl;
        }
        Final_Commit<<<b_per_grid,thrd_blck>>>(d_c, d_k, node_size);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            std::cout<<"Error synchronizing device"<<endl;
        }
        if(!HandleCUDAError(hipEventRecord(stop))){
            std::cout<<"Error recording stop event"<<endl;
        }
        if(!HandleCUDAError(hipEventSynchronize(stop))){
            std::cout<<"Error synchronizing stop event"<<endl;
        }
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout<<"Time elapsed FrogWild: "<<milliseconds<<" ms"<<endl;
        if(!HandleCUDAError(hipMemcpy(global_src, d_global_src, (node_size+1)*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to global_src"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(global_succ, d_global_succ, (edge_size)*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to global_succ"<<endl;
        }
        
        hipFree(d_unq);
        hipFree(d_unq_ptr);
        hipFree(d_replica);
        hipFree(d_p_t);
        hipFree(d_p_s);
        hipFree(local_K);
        hipFree(local_C);
        hipFree(d_global_src);
        hipFree(d_global_succ);
        hipFree(d_state_teleport);
        hipFree(d_state_scatter);
        hipFree(rand_frog);
        thrust::sequence(ind_rank, ind_rank+node_size,1);
        unsigned int* dev_ind_ptr_approx;
        if(!HandleCUDAError(hipMalloc((void**)&dev_ind_ptr_approx, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for dev_ind_ptr_approx"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(dev_ind_ptr_approx, ind_rank, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to dev_ind_ptr_approx"<<endl;
        }
        thrust::stable_sort_by_key(thrust::device,d_c, d_c+node_size, dev_ind_ptr_approx, thrust::greater<float>());
        if(!HandleCUDAError(hipMemcpy(ind_rank, dev_ind_ptr_approx, node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to h_indices_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(c, d_c, node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to c"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(k, d_k, node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to k"<<endl;
        }
        hipFree(d_c);
        hipFree(d_k);
        if(!HandleCUDAError(hipMemGetInfo( &free_byte, &total_byte ))){
            std::cout<<"Error getting memory info"<<endl;
        }
        free_db = (double)free_byte ;
        total_db = (double)total_byte ;
        used_db = total_db - free_db ;
        printf("GPU memory usage before PR: used = %f, free = %f MB, total = %f MB\n",
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
        //Perform PageRank with cuSparse and cuBLAS
        std::cout<<"Performing PageRank"<<endl;
        float* pagerank;
        pagerank = new float[node_size]; 
        unsigned int *indices;
        indices = new unsigned int[node_size];
        thrust::sequence(indices, indices+node_size,1);
        unsigned int max_iter = iter;
        float tol = 1e-14;   
        float damp = p_t;
        PageRank(pagerank,indices, global_src, global_succ, damp, node_size, edge_size, max_iter, tol);
        /*We need to do accuracy stuff here, for now, we need to verify with python*/

        Export_pr_vector(pagerank,indices, node_size);
        delete[] pagerank;
        delete[] indices;
    }
    else if(version==1){
        if(!HandleCUDAError(hipMalloc((void**)&d_unq, (unq_ptr[BLOCKS])*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_unq"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_c, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_c"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_k, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_k"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_unq_ptr, (BLOCKS+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_unq_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_replica, node_size*sizeof(replica_tracker)))){
            std::cout<<"Error allocating memory for d_replica"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_p_t, sizeof(float)))){
            std::cout<<"Error allocating memory for d_p_t"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_p_s, sizeof(float)))){
            std::cout<<"Error allocating memory for d_p_s"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&num_local_C, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for num_local_K"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&num_local_K, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for num_local_K"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_K, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_K"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_C, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_C"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_K_idx, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_K_idx"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_global_src, (node_size+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_global_src"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_global_succ, (edge_size)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_global_succ"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_unq, unq, (unq_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_unq"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_unq_ptr, unq_ptr, (BLOCKS+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_unq_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_replica, h_replica, node_size*sizeof(replica_tracker), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_replica"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_p_t, &p_t, sizeof(float), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_p_t"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_p_s, &p_s, sizeof(float), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_p_s"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_global_src, global_src, (node_size+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_global_src"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_global_succ, global_succ, (edge_size)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_global_succ"<<endl;
        }
        float* rand_frog;
        int sublinear_size=node_size/8;
        if(!HandleCUDAError(hipMalloc((void**)&rand_frog, sublinear_size*sizeof(float)))){
            std::cout<<"Error allocating memory for rand_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemset(rand_frog, 0, sublinear_size*sizeof(float)))){
            std::cout<<"Error initializing rand_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemset(d_k, 0, node_size*sizeof(unsigned int)))){
            std::cout<<"Error initializing d_k"<<endl;
        }
        if(!HandleCUDAError(hipMemset(d_c, 0, node_size*sizeof(unsigned int)))){
            std::cout<<"Error initializing d_c"<<endl;
        }
        if(!HandleCUDAError(hipMemset(num_local_C, 0, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error initializing num_local_C"<<endl;
        }
        if(!HandleCUDAError(hipMemset(num_local_K, 0, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error initializing num_local_K"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_K, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_K"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_C, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_C"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_K_idx, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_K_idx"<<endl;
        }
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, rand_frog, sublinear_size);
        /*Now, we have the random numbers generated*/
        hiprandDestroyGenerator(gen);
        unsigned int t_per_block = thrd_blck;
        unsigned int b_per_grid_int = (sublinear_size+thrd_blck-1)/thrd_blck;
        unsigned int b_per_grid = (node_size+thrd_blck-1)/thrd_blck;
        hiprandState* d_state_teleport;
        if(!HandleCUDAError(hipMalloc((void**)&d_state_teleport, BLOCKS*thrd_blck*sizeof(hiprandState)))){
            std::cout<<"Error allocating memory for d_state"<<endl;
        }
        hiprandState* d_state_scatter;
        if(!HandleCUDAError(hipMalloc((void**)&d_state_scatter, BLOCKS*thrd_blck*sizeof(hiprandState)))){
            std::cout<<"Error allocating memory for d_state"<<endl;
        }
        First_Init<<<b_per_grid_int, t_per_block>>>(rand_frog, d_k, node_size, sublinear_size);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            std::cout<<"Error synchronizing device"<<endl;
        }
        hipEvent_t start, stop;
        if(!HandleCUDAError(hipEventCreate(&start))){
            std::cout<<"Error creating start event"<<endl;
        }
        if(!HandleCUDAError(hipEventCreate(&stop))){
            std::cout<<"Error creating stop event"<<endl;
        }
        if(!HandleCUDAError(hipEventRecord(start))){
            std::cout<<"Error recording start event"<<endl;
        }
        hipFuncSetAttribute(reinterpret_cast<const void*>(Apply_Ver0), hipFuncAttributeMaxDynamicSharedMemorySize, 102400);
        // for(unsigned int i=0; i<iter; i++){
        //     std::cout<<"Iteration "<<i<<endl;
        //     Gather_Ver1<<<BLOCKS,thrd_blck>>>(d_k, d_unq, d_unq_ptr, num_local_K, local_K, local_K_idx);
        //     if(!HandleCUDAError(hipDeviceSynchronize())){
        //         std::cout<<"Error synchronizing device"<<endl;
        //     }
        //     std::cout<<"Gathered"<<endl;
        //     std::cout<<max_unq_ctr*sizeof(unsigned int)<<endl;
        //     Apply_Ver1<<<BLOCKS, thrd_blck, max_unq_ctr*sizeof(unsigned int)>>>(d_unq_ptr, local_K, local_C,num_local_K,local_K_idx, d_p_t,i, d_state_teleport);
        //     if(!HandleCUDAError(hipDeviceSynchronize())){
        //         std::cout<<"Error synchronizing device for Apply"<<endl;
        //     }
        //     std::cout<<"Applied"<<endl;
        //     Sync_Mirrors_Ver1<<<BLOCKS,thrd_blck>>>(d_c, d_k, d_unq, d_unq_ptr, local_C, local_K, local_C_idx, local_K_idx, num_local_C, num_local_K, d_p_s, d_state_scatter);
        //     if(!HandleCUDAError(hipDeviceSynchronize())){
        //         std::cout<<"Error synchronizing device for Sync"<<endl;
        //     }
        //     std::cout<<"Synced"<<endl;
        //     Scatter_Ver1<<<b_per_grid,thrd_blck>>>(d_c, d_k, d_global_src, d_global_succ, d_replica, node_size);
        //     if(!HandleCUDAError(hipDeviceSynchronize())){
        //         std::cout<<"Error synchronizing device for Scatter"<<endl;
        //     }
        //     std::cout<<"Scattered"<<endl;
        //     if(!HandleCUDAError(hipMemset(num_local_K, 0, BLOCKS*sizeof(unsigned int)))){
        //         std::cout<<"Error rewriting num of local K"<<endl;
        //     }
        //     if(!HandleCUDAError(hipMemset(num_local_C, 0, BLOCKS*sizeof(unsigned int)))){
        //         std::cout<<"Error rewriting num of local C"<<endl;
        //     }
        //     if(!HandleCUDAError(hipMemset(local_K, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
        //         std::cout<<"Error rewriting local K"<<endl;
        //     }
        //     if(!HandleCUDAError(hipMemset(local_C, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
        //         std::cout<<"Error rewriting local C"<<endl;
        //     }
        //     if(!HandleCUDAError(hipMemset(local_K_idx, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
        //         std::cout<<"Error rewriting local K idx"<<endl;
        //     }
        // }
        if(!HandleCUDAError(hipEventRecord(stop))){
            std::cout<<"Error recording stop event"<<endl;
        }
        if(!HandleCUDAError(hipEventSynchronize(stop))){
            std::cout<<"Error synchronizing stop event"<<endl;
        }
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        std::cout<<"Time elapsed FrogWild: "<<milliseconds<<" ms"<<endl;
        Final_Commit<<<BLOCKS,thrd_blck>>>(d_c, d_k, node_size);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            std::cout<<"Error synchronizing device"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(global_src, d_global_src, (node_size+1)*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to global_src"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(global_succ, d_global_succ, (edge_size)*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to global_succ"<<endl;
        }
        
        hipFree(d_unq);
        hipFree(d_unq_ptr);
        hipFree(d_replica);
        hipFree(d_p_t);
        hipFree(d_p_s);
        hipFree(num_local_K);
        hipFree(num_local_C);
        hipFree(local_K);
        hipFree(local_C);
        hipFree(local_K_idx);
        hipFree(d_global_src);
        hipFree(d_global_succ);
        hipFree(d_state_teleport);
        hipFree(d_state_scatter);
        hipFree(rand_frog);
        //Perform PageRank with cuSparse and cuBLAS
        std::cout<<"Performing PageRank"<<endl;
        float* pagerank;
        pagerank = new float[node_size]; 
        unsigned int *indices, *indices_approx;
        indices = new unsigned int[node_size];
        indices_approx = new unsigned int[node_size];
        unsigned int* dev_ind_ptr_approx;
        if(!HandleCUDAError(hipMalloc((void**)&dev_ind_ptr_approx, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for dev_ind_ptr_approx"<<endl;
        }
        thrust::sequence(indices, indices+node_size,1);
        thrust::sequence(indices_approx, indices_approx+node_size,1);
        unsigned int max_iter = 100;
        float tol = 1e-14;   
        float damp = p_t;
        PageRank(pagerank,indices, global_src, global_succ, damp, node_size, edge_size, max_iter, tol);
        /*We need to do accuracy stuff here, for now, we need to verify with python*/

        Export_pr_vector(pagerank,indices, node_size);
        thrust::stable_sort_by_key(thrust::device,d_c, d_c+node_size, dev_ind_ptr_approx, thrust::greater<float>());
        if(!HandleCUDAError(hipMemcpy(ind_rank, dev_ind_ptr_approx, node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to h_indices_frog"<<endl;
        }
        delete[] pagerank;
        delete[] indices;
        delete[] indices_approx;
        if(!HandleCUDAError(hipMemcpy(c, d_c, node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to c"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(k, d_k, node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
            std::cout<<"Error copying memory to k"<<endl;
        }
        hipFree(d_c);
        hipFree(d_k);

    }

    else{
        if(!HandleCUDAError(hipMalloc((void**)&d_succ, (h_ptr[BLOCKS])*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_succ"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_src, (src_ptr[BLOCKS])*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_src"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_unq, (unq_ptr[BLOCKS])*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_unq"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_c, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_c"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_k, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_k"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_src_ptr, (BLOCKS+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_src_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_unq_ptr, (BLOCKS+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_unq_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_h_ptr, (BLOCKS+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_h_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_degree, node_size*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_degree"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_replica, node_size*sizeof(replica_tracker)))){
            std::cout<<"Error allocating memory for d_replica"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_p_t, sizeof(float)))){
            std::cout<<"Error allocating memory for d_p_t"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_p_s, sizeof(float)))){
            std::cout<<"Error allocating memory for d_p_s"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&num_local_K, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for num_local_K"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&num_local_C, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for num_local_C"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_K, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_K"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_C, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_C"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_K_idx, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_K_idx"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&local_C_idx, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for local_C_idx"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_global_src, (node_size+1)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_global_src"<<endl;
        }
        if(!HandleCUDAError(hipMalloc((void**)&d_global_succ, (edge_size)*sizeof(unsigned int)))){
            std::cout<<"Error allocating memory for d_global_succ"<<endl;
        }
        std::cout<<"Copying memory to device variables"<<endl;
        if(!HandleCUDAError(hipMemcpy(d_succ, local_succ, (h_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_succ"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_src, local_src, (src_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_src"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_unq, unq, (unq_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_unq"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_c, c, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_c"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_k, k, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_k"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_src_ptr, src_ptr, (BLOCKS+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_src_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_unq_ptr, unq_ptr, (BLOCKS+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_unq_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_h_ptr, h_ptr, (BLOCKS+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_h_ptr"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_degree, degree, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_degree"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_replica, h_replica, node_size*sizeof(replica_tracker), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_replica"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_p_t, &p_t, sizeof(float), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_p_t"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_p_s, &p_s, sizeof(float), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_p_s"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_global_src, global_src, (node_size+1)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_global_src"<<endl;
        }
        if(!HandleCUDAError(hipMemcpy(d_global_succ, global_succ, (edge_size)*sizeof(unsigned int), hipMemcpyHostToDevice))){
            std::cout<<"Error copying memory to d_global_succ"<<endl;
        }
        float* rand_frog;
        int sublinear_size=node_size/8;
        if(!HandleCUDAError(hipMalloc((void**)&rand_frog, sublinear_size*sizeof(float)))){
            std::cout<<"Error allocating memory for rand_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemset(rand_frog, 0, sublinear_size*sizeof(float)))){
            std::cout<<"Error initializing rand_frog"<<endl;
        }
        if(!HandleCUDAError(hipMemset(d_k, 0, node_size*sizeof(unsigned int)))){
            std::cout<<"Error initializing d_k"<<endl;
        }
        if(!HandleCUDAError(hipMemset(d_c, 0, node_size*sizeof(unsigned int)))){
            std::cout<<"Error initializing d_c"<<endl;
        }
        if(!HandleCUDAError(hipMemset(num_local_C, 0, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error initializing num_local_C"<<endl;
        }
        if(!HandleCUDAError(hipMemset(num_local_K, 0, BLOCKS*sizeof(unsigned int)))){
            std::cout<<"Error initializing num_local_K"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_K, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_K"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_C, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_C"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_K_idx, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_K_idx"<<endl;
        }
        if(!HandleCUDAError(hipMemset(local_C_idx, 0, unq_ptr[BLOCKS]*sizeof(unsigned int)))){
            std::cout<<"Error initializing local_C_idx"<<endl;
        }
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, rand_frog, sublinear_size);
        /*Now, we have the random numbers generated*/
        hiprandDestroyGenerator(gen);
        unsigned int t_per_block = thrd_blck;
        unsigned int b_per_grid_int = (sublinear_size+thrd_blck-1)/thrd_blck;
        hiprandState* d_state_teleport;
        if(!HandleCUDAError(hipMalloc((void**)&d_state_teleport, BLOCKS*thrd_blck*sizeof(hiprandState)))){
            std::cout<<"Error allocating memory for d_state"<<endl;
        }
        hiprandState* d_state_scatter;
        if(!HandleCUDAError(hipMalloc((void**)&d_state_scatter, BLOCKS*thrd_blck*sizeof(hiprandState)))){
            std::cout<<"Error allocating memory for d_state"<<endl;
        }
        First_Init<<<b_per_grid_int, t_per_block>>>(rand_frog, d_k, node_size, sublinear_size);
        if(!HandleCUDAError(hipDeviceSynchronize())){
            std::cout<<"Error synchronizing device"<<endl;
        }
        hipFree(d_succ);
        hipFree(d_src);
        hipFree(d_unq);
        hipFree(d_c);
        hipFree(d_k);
        hipFree(d_src_ptr);
        hipFree(d_unq_ptr);
        hipFree(d_h_ptr);
        hipFree(d_degree);
        hipFree(d_replica);
        hipFree(d_p_t);
        hipFree(d_p_s);
        hipFree(num_local_K);
        hipFree(num_local_C);
        hipFree(local_K);
        hipFree(local_C);
        hipFree(local_K_idx);
        hipFree(local_C_idx);
        hipFree(d_global_src);
        hipFree(d_global_succ);
        hipFree(d_state_teleport);
        hipFree(d_state_scatter);
        hipFree(rand_frog);
    }

}



__global__ void First_Init(float* rand_frog, unsigned int* K, unsigned int node_size, unsigned int sublinear_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    if(idx<sublinear_size){
        rand_frog[idx]=floorf(rand_frog[idx]*node_size);
        if(rand_frog[idx]<node_size){
            atomicAdd(&K[(unsigned int)rand_frog[idx]],1);
        }
        else{
            atomicAdd(&K[(unsigned int)rand_frog[idx]%node_size],1);
        }
    }
}


/*
What we need for the iterations of pagerank:
(1)Gather
(2)Apply
(3)Scatter
---------------------------------------------
(1) Gather: 
-First time, initialize random frogs (done)
-Remaining iterations, we need to collect the frogs from the previous iteration sent to nodes from scatter

(2) Apply:
-This function takes care for keeping track of the number of frogs that have stopped on each vertex

(3)Scatter 
-This function takes care of sending the frogs to the next vertex

Instead of dictating which block is the master of which vertex, we will have the global memory act as the sole master
of the vertex. This will allow us to combine the functions into one and avoid passing of data, and ease the synchronization
*/

__global__ void Gather_Ver0(unsigned int* K, unsigned int* unq, unsigned int* unq_ptr,unsigned int* local_K){
    unsigned int tid = threadIdx.x;
    const unsigned int len_nodes_clust=unq_ptr[blockIdx.x+1]-unq_ptr[blockIdx.x];
    const unsigned int c_v_len = len_nodes_clust/blockDim.x+1;
    for(int i=tid; i<len_nodes_clust; i+=blockDim.x){
        //unq contains the unqiue nodes in the cluster
        //unq_ptr contains the pointers to the start of each cluster
        //Hence referencing unq[i+unq_ptr[blockIdx.x]] will give the node in the cluster, pointing to K
        //This is the node that we are going to be looking at
        if(K[unq[i+unq_ptr[blockIdx.x]]]>0){
            *(local_K+unq_ptr[blockIdx.x]+i)=K[unq[i+unq_ptr[blockIdx.x]]];
            //We are going to have replicas of frogs as well, additional care/attention should be made for handling this
            //Do we naively divide the count at the end by the number of replicas if there are going to be mulitplicities?
            //Possibly a question worth experimentation
            //Local_K_idx is going to store the index of the unique value with a frog on it
        }
        __syncthreads();
    }
    /*To summarize what has been done here
    (1), we increment the value of the number of non zero k values, i.e. we are using an array to identify how many vertices should be active in the
    next function so as to avoid warp divergence
    (2) As we increment the number of non zero K values, we use the new value as a memory pointer to identify that we need to place a new
    value in the next memory location
    (2a) Using num_local_K, we then store the K value in local_K, pointing the the block and then the offset based on the current num_local_K
    (2b) We then save the global address of K in local_K_idx*/
}


__global__ void Apply_Ver0(unsigned int* unq_ptr, unsigned int* local_K_global,unsigned int* local_K_temp, unsigned int* local_C_global, float* p_t, unsigned int iter, hiprandState* d_state){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    const unsigned int len_nodes_clust=unq_ptr[blockIdx.x+1]-unq_ptr[blockIdx.x];
    unsigned int* local_K = local_K_temp+unq_ptr[blockIdx.x];
    for(int i=tid; i<len_nodes_clust; i+=blockDim.x){
        local_K[i]=local_K_global[i+unq_ptr[blockIdx.x]];
    }
    __syncthreads();
    // if(tid==0)
    //     printf("Block %d is done with copying\n",blockIdx.x);
    for(unsigned int i=tid; i<len_nodes_clust; i+=blockDim.x){
        //This loop iterates throught the unique vertex values in a block
        for(int j=0; j<*(local_K_global+unq_ptr[blockIdx.x]+i); j++){
            //This loop iterates through the number of living frogs on a vertex
            hiprand_init(1234+j+iter, idx, 0, &d_state[idx]);
            float rand = hiprand_uniform(&d_state[idx]);
            //The above section is to generate a random number for each frog
            //The index doing this seems as if it will have the same random
            //number for each frog, so incrementing the seed by j should (in theory)
            //give each frog a unique random number
            if(rand<*(p_t)){
                atomicAdd(local_C_global+unq_ptr[blockIdx.x]+i,1);
                //Increment the number of frogs which have died on this vertex-this will mirror the indexing of the unq ptr
                // atomicAdd(num_loc_C+blockIdx.x,1);
                // //Increment the number of non zero C values
                // *(local_C_idx+unq_ptr[blockIdx.x]+*(num_loc_C+blockIdx.x))=*(local_K_idx+unq_ptr[blockIdx.x]+tid);
                //The local C index in this block is going to be the same as the local K index
                //Notice that we are using the number of non-zero C's for this
                //The issue with the above part could exceed the values, this poses an issue- do we need this?
                //I do not think so
                atomicSub(local_K+i,1);
                //Decrement the K value
            }
        }
    }
    __syncthreads();

    // if(tid==0){
    //     printf("BLock %d is done with iterating\n",blockIdx.x);
    // }
    for(int i=tid; i<len_nodes_clust; i+=blockDim.x){
        local_K_global[i+unq_ptr[blockIdx.x]]=local_K[i];
    }

    //This tells which vertices have frogs that have stopped
    // __syncthreads();
}

__global__ void Sync_Mirrors_Ver0(unsigned int* C, unsigned int* K, unsigned int* unq, unsigned int* unq_ptr, unsigned int* local_C, unsigned int* local_K, 
unsigned int* src, unsigned int* succ, unsigned int* mirror_ctr,replica_tracker* d_rep, unsigned int node_size, float* p_s, hiprandState* d_state){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    const unsigned int len_nodes_clust=unq_ptr[blockIdx.x+1]-unq_ptr[blockIdx.x];
    hiprand_init(1234, idx, 0, &d_state[idx]);
    //We have this outside so if the if condition is satisfied, the entirety of local C can be committed
    //to the global C
    float rand = hiprand_uniform(&d_state[idx]);
    for(int i=tid; i<len_nodes_clust; i+=blockDim.x){
        if(rand<*(p_s)){
            atomicAdd(mirror_ctr+unq[i+unq_ptr[blockIdx.x]],1);
            for(int j=0; j<local_C[unq_ptr[blockIdx.x]+i]; j++){
                //Commit to global memory
                    atomicAdd(C+unq[i+unq_ptr[blockIdx.x]],1);
            }
        }
    }
    for(int i=idx; i<node_size; i+=gridDim.x*blockDim.x){
        if(K[i]>0 && rand<*(p_s)){
            unsigned int num_frog=(mirror_ctr[i]>0)?(K[i]/(mirror_ctr[i])+1):(0);
            // printf("Im going to catch %u frogs\n",num_frog);
            // printf("I am vertex %u\n",i);
            // printf("I have %u replicas\n",d_rep[i].num_replicas);
            for(int j=src[i]; j<src[i+1]; j++){
                atomicAdd(&K[succ[j]],num_frog);
                // K[i]-=(K[i]>num_frog)?(num_frog):(K[i]);
            }
        }
    }
}

__global__ void Scatter_Ver0(unsigned int* C, unsigned int* K, unsigned int* src, unsigned int* succ,replica_tracker* d_rep, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    for(int i=idx; i<node_size; i+=gridDim.x*blockDim.x){
        if(K[i]>0){
            unsigned int num_frog=K[i]/d_rep[i].num_replicas+1;
            // printf("Im going to catch %u frogs\n",num_frog);
            // printf("I am vertex %u\n",i);
            // printf("I have %u replicas\n",d_rep[i].num_replicas);
            for(int j=src[i]; j<src[i+1]; j++){
                atomicAdd(&K[succ[j]],num_frog);
                // K[i]-=(K[i]>num_frog)?(num_frog):(K[i]);
            }
        }
    }
}

__global__ void Final_Commit(unsigned int* C, unsigned int* K, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    for(int i=idx; i<node_size; i+=gridDim.x*blockDim.x){
        C[i]+=K[i];
    }
    __syncthreads();
}

__global__ void Reverse_Gather(unsigned int* K, unsigned int* local_K, replica_tracker* d_rep, unsigned int* unq, unsigned int* unq_ptr, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int len_nodes_clust=unq_ptr[blockIdx.x+1]-unq_ptr[blockIdx.x];
    for(unsigned int i =tid; i<len_nodes_clust; i+=blockDim.x){
        if(local_K[i]>0){
            unsigned int num_frog=local_K[i]/d_rep[unq[i+unq_ptr[blockIdx.x]]].num_replicas+1;
            for(int j=0; j<num_frog; j++){
                atomicAdd(&K[unq[i+unq_ptr[blockIdx.x]]],1);
            }
        }
        __syncthreads();
    }
}

//Thoughts- maybe save multiple files of the number of nodes and commit to them with the C to sync with the mirrors

//There should be another better way


__global__ void Gather_Ver1(unsigned int* K, unsigned int* unq, unsigned int* unq_ptr, unsigned int* num_local_K,
unsigned int* local_K, unsigned int* local_K_idx){
    unsigned int tid = threadIdx.x;
    const unsigned int len_nodes_clust=unq_ptr[blockIdx.x+1]-unq_ptr[blockIdx.x];
    const unsigned int c_v_len = len_nodes_clust/blockDim.x+1;
    for(int i=tid; i<len_nodes_clust; i+=blockDim.x){
        //unq contains the unqiue nodes in the cluster
        //unq_ptr contains the pointers to the start of each cluster
        //Hence referencing unq[i+unq_ptr[blockIdx.x]] will give the node in the cluster, pointing to K
        //This is the node that we are going to be looking at
        if(K[unq[i+unq_ptr[blockIdx.x]]]>0){
            atomicAdd(num_local_K+blockIdx.x,1);
            atomicExch(local_K+unq_ptr[blockIdx.x]+num_local_K[blockIdx.x]-1,K[unq[i+unq_ptr[blockIdx.x]]]);
            atomicExch(local_K_idx+unq_ptr[blockIdx.x]+num_local_K[blockIdx.x]-1,i);
            //We are going to have replicas of frogs as well, additional care/attention should be made for handling this
            //Do we naively divide the count at the end by the number of replicas if there are going to be mulitplicities?
            //Possibly a question worth experimentation
            //Local_K_idx is going to store the index of the unique value with a frog on it
        }
        __syncthreads();
    }
    /*To summarize what has been done here
    (1), we increment the value of the number of non zero k values, i.e. we are using an array to identify how many vertices should be active in the
    next function so as to avoid warp divergence
    (2) As we increment the number of non zero K values, we use the new value as a memory pointer to identify that we need to place a new
    value in the next memory location
    (2a) Using num_local_K, we then store the K value in local_K, pointing the the block and then the offset based on the current num_local_K
    (2b) We then save the global address of K in local_K_idx*/
}


__global__ void Apply_Ver1(unsigned int* unq_ptr, unsigned int* K, unsigned int* C, unsigned int* num_loc_K, unsigned int* local_K_idx, float* p_t, hiprandState* d_state){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ unsigned int local_K[];
    for(int i = tid; i<num_loc_K[blockIdx.x]; i+=blockDim.x){
        local_K[i]=K[unq_ptr[blockIdx.x]+i];
    }
    for(unsigned int i=tid; i<num_loc_K[blockIdx.x]; i+=blockDim.x){
        for(int j=0; j<*(K+unq_ptr[blockIdx.x]+i); j++){
            hiprand_init(1234+j, idx, 0, &d_state[idx]);
            float rand = hiprand_uniform(&d_state[idx]);
            //The above section is to generate a random number for each frog
            //The index doing this seems as if it will have the same random
            //number for each frog, so incrementing the seed by j should (in theory)
            //give each frog a unique random number
            if(rand<*(p_t)){
                atomicAdd(C+unq_ptr[blockIdx.x]+local_K_idx[i],1);
                //Increment the number of frogs which have died on this vertex-this will mirror the indexing of the unq ptr
                // atomicAdd(num_loc_C+blockIdx.x,1);
                // //Increment the number of non zero C values
                // *(local_C_idx+unq_ptr[blockIdx.x]+*(num_loc_C+blockIdx.x))=*(local_K_idx+unq_ptr[blockIdx.x]+tid);
                //The local C index in this block is going to be the same as the local K index
                //Notice that we are using the number of non-zero C's for this
                //The issue with the above part could exceed the values, this poses an issue- do we need this?
                //I do not think so
                atomicSub(local_K+i,1);
                //Decrement the K value
                if(local_K[i]==0){
                    atomicSub(num_loc_K+blockIdx.x,1);
                }
            }
        }
    }

    //This tells which vertices have frogs that have stopped
    __syncthreads();
}

__global__ void Sync_Mirrors_Ver1(unsigned int* C, unsigned int* K, unsigned int* unq, unsigned int* unq_ptr, unsigned int* local_C, 
unsigned int* local_K, float* p_s, replica_tracker* d_replica, hiprandState* d_state){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    const unsigned int len_nodes_clust=unq_ptr[blockIdx.x+1]-unq_ptr[blockIdx.x];
    hiprand_init(1234, idx, 0, &d_state[idx]);
    //We have this outside so if the if condition is satisfied, the entirety of local C can be committed
    //to the global C
    float rand = hiprand_uniform(&d_state[idx]);
    for(unsigned int i =tid; i<len_nodes_clust; i+=blockDim.x){
        for(int j=0; j<*(local_C+unq_ptr[blockIdx.x]+i); j++){
            //Commit to global memory
            if(rand<*(p_s) && *(local_C+unq_ptr[blockIdx.x]+i)>0){
                atomicAdd(C+unq[i+unq_ptr[blockIdx.x]],1);
                *(local_C+unq_ptr[blockIdx.x]+i)-=1;
            }
        }
        __syncthreads();
    }
}

__global__ void Scatter_Ver1(unsigned int* C, unsigned int* K, unsigned int* src, unsigned int* succ,replica_tracker* d_rep, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    if(idx<node_size){
        if(K[idx]>0){
            unsigned int num_frog=K[idx]/d_rep[idx].num_replicas+1;
            for(int i=src[idx]; i<src[idx+1]; i++){
                atomicAdd(&K[succ[i]],num_frog);
            }
        }
    }
}


__global__ void Schur_Product_Vectors(unsigned int* vect_1, unsigned int* vect_2, unsigned int* res_vec, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<size){
        res_vec[idx]=vect_1[idx]*vect_2[idx];
    }
}

__global__ void Partial_Sums(unsigned int* res_vec, unsigned int* last_val, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int temp=0;
    if (idx>=size){
        return;
    }
    unsigned int* blockAddress= res_vec+(blockDim.x*blockIdx.x);
    for(int stride=blockDim.x/2; stride>0; stride>>=1){
        if (tid<stride && tid+stride<size){
            blockAddress[tid]+=blockAddress[tid+stride];
        }
        __syncthreads();
    }
    if(tid==0){
        last_val[blockIdx.x]=blockAddress[0];
    }
}

__global__ void Compute_L2_Max_u_1(unsigned int* vect_1, unsigned int* res_vec_1, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx<size){
        res_vec_1[idx]=vect_1[idx]*vect_1[idx];
    }
}


__global__ void Partial_Sum_Last_Val(unsigned int* last_val, unsigned int* res, unsigned int block_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int* blockAddress=last_val;
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
	{
		if (tid < stride)
		{
			//tid<stride ensures we do not try to access memory past the vector allocated to the block
			//tid+stride<size allows for vector sizes less than blockDim
			blockAddress[tid] += blockAddress[tid + stride];
		}
		__syncthreads();//Make all of the threads wait to go to the next iteration so the values are up to date
	}
    if(tid==0){
        *(res)=(blockAddress[0]);
    }

}


__host__ void Verif_Dot_Product(unsigned int* vec_1, unsigned int* vec_2, unsigned int res, unsigned int size){
    unsigned int temp=0;
    for(int i=0; i<size; i++){
        temp+=vec_1[i]*vec_2[i];
    }
    if(temp==res){
        std::cout<<"Dot product is correct"<<endl;
    }
    else{
        std::cout<<"Dot product is incorrect"<<endl;
        std::cout<<"GPU "<<res<<endl;
        std::cout<<"CPU "<<temp<<endl;
    }
}

__host__ void Verif_L2(unsigned int* vec, unsigned int res, unsigned int size){
    unsigned int temp=0;
    for(int i=0; i<size; i++){
        temp+=vec[i]*vec[i];
    }
    if(temp==res){
        std::cout<<"L2 norm is correct"<<endl;
    }
    else{
        std::cout<<"L2 norm is incorrect"<<endl;
        std::cout<<"GPU "<<res<<endl;
        std::cout<<"CPU "<<temp<<endl;
    }
} 