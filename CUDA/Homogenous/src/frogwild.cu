#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"


__host__ void Import_Local_Src(unsigned int* local_src){
    ifstream myfile;
    myfile.open(LOCAL_SRC_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        local_src[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}


__host__ void Import_Local_Succ(unsigned int* local_succ){
    ifstream myfile;
    myfile.open(LOCAL_SUCC_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        local_succ[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}

__host__ void Import_Unique(unsigned int* unq){
    ifstream myfile;
    myfile.open(UNQ_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        unq[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}



__host__ void Import_Src_Ctr_Ptr(unsigned int* src_ctr, unsigned int* src_ptr){
    ifstream myfile;
    myfile.open(SRC_CTR_PTR_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        src_ctr[count-1] = stoi(word);
                        column++;
                    }
                    else{
                        src_ptr[count-1] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }

}


__host__ void Import_Unq_Ptr_Ctr(unsigned int* unq_ptr, unsigned int* unq_ctr){
    ifstream myfile;
    myfile.open(UNQ_CTR_PTR_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        unq_ctr[count-1] = stoi(word);
                        column++;
                    }
                    else{
                        unq_ptr[count-1] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }

}


__host__ void Import_H_Ctr_Ptr(unsigned int* h_ctr, unsigned int* h_ptr){
    ifstream myfile;
    myfile.open(H_CTR_PTR_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        h_ctr[count-1] = stoi(word);
                        column++;
                    }
                    else{
                        h_ptr[count-1] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }

}


__host__ void Import_Degree(unsigned int* deg, unsigned int node_size){
    ifstream myfile;
    myfile.open(DEG_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else{
                        deg[count-1] = stoi(word);
                    }
                }
            }
            column = 0;
            count++;
        }
    }
}

__host__ void Import_Replica_Stats(replica_tracker* h_replica, unsigned int node_size){
    ifstream myfile;
    myfile.open(REPLICA_STAT_PATH);
    string line,word;
    int count = 0;
    int column = 0;
    if(!myfile.is_open()){
        cout << "Error opening file" << endl;
        exit(1);
    }
    else{
        while(getline(myfile,line)){
            stringstream s(line);
            while(getline(s,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        column++;
                    }
                    else if(column==1){
                        h_replica[count-1].num_replicas = stoi(word);
                        column++;
                    }
                    else{
                        h_replica[count-1].clusters[column-2] = stoi(word);
                    }
                }
            }
            count++;
            column = 0;
        }
    }
}

__host__ void FrogWild(unsigned int* local_succ, unsigned int* local_src, unsigned int* unq, unsigned int* c, unsigned int* k, unsigned int* src_ptr, 
unsigned int* unq_ptr, unsigned int* h_ptr, unsigned int* degree, replica_tracker* h_replica, int node_size, unsigned int edge_size){
    unsigned int *d_succ, *d_src, *d_unq, *d_c, *d_k, *d_src_ptr, *d_unq_ptr, *d_h_ptr, *d_degree;
    replica_tracker *d_replica;
    if(!HandleCUDAError(hipMalloc((void**)&d_succ, (h_ptr[BLOCKS])*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_src, (src_ptr[BLOCKS])*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_src"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_unq, (unq_ptr[BLOCKS])*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_unq"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_c, node_size*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_c"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_k, node_size*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_k"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_src_ptr, BLOCKS*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_src_ptr"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_unq_ptr, BLOCKS*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_unq_ptr"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_h_ptr, BLOCKS*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_h_ptr"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_degree, node_size*sizeof(unsigned int)))){
        cout<<"Error allocating memory for d_degree"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_replica, node_size*sizeof(replica_tracker)))){
        cout<<"Error allocating memory for d_replica"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_succ, local_succ, (h_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_succ"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_src, local_src, (src_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_src"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_unq, unq, (unq_ptr[BLOCKS])*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_unq"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_c, c, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_c"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_k, k, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_k"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_src_ptr, src_ptr, BLOCKS*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_src_ptr"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_unq_ptr, unq_ptr, BLOCKS*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_unq_ptr"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_h_ptr, h_ptr, BLOCKS*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_h_ptr"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_degree, degree, node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_degree"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_replica, h_replica, node_size*sizeof(replica_tracker), hipMemcpyHostToDevice))){
        cout<<"Error copying memory to d_replica"<<endl;
    }
    /*Now, all of the memory has been transferred and allocated*/
    /*Generate a float vector to hold the random numbers for this first intialization*/
    float* rand_frog;
    int sublinear_size=node_size/8;
    if(!HandleCUDAError(hipMalloc((void**)&rand_frog, sublinear_size*sizeof(float)))){
        cout<<"Error allocating memory for rand_frog"<<endl;
    }
    if(!HandleCUDAError(hipMemset(rand_frog, 0, sublinear_size*sizeof(float)))){
        cout<<"Error initializing rand_frog"<<endl;
    }
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateUniform(gen, rand_frog, sublinear_size);
    /*Now, we have the random numbers generated*/
    hiprandDestroyGenerator(gen);
    unsigned int t_per_block = TPB;
    unsigned int b_per_grid_int = (sublinear_size+TPB-1)/TPB;
    First_Init<<<b_per_grid_int, t_per_block>>>(rand_frog, d_k, node_size, sublinear_size);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Error synchronizing device"<<endl;
    }


}



__global__ void First_Init(float* rand_frog, unsigned int* K, unsigned int node_size, unsigned int sublinear_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    if(idx<sublinear_size){
        rand_frog[idx]=floorf(rand_frog[idx]*node_size);
        if(rand_frog[idx]<node_size){
            atomicAdd(&K[(unsigned int)rand_frog[idx]],1);
        }
        else{
            atomicAdd(&K[(unsigned int)rand_frog[idx]%node_size],1);
        }
    }
}


/*
What we need for the iterations of pagerank:
(1)Gather
(2)Apply
(3)Scatter
---------------------------------------------
(1) Gather: 
-First time, initialize random frogs (done)
-Remaining iterations, we need to collect the frogs from the previous iteration sent to nodes from scatter

(2) Apply:
-This function takes care for keeping track of the number of frogs that have stopped on each vertex

(3)Scatter 
-This function takes care of sending the frogs to the next vertex

Instead of dictating which block is the master of which vertex, we will have the global memory act as the sole master
of the vertex. This will allow us to combine the functions into one and avoid passing of data, and ease the synchronization
*/

__global__ void Apply(unsigned int* local_src, unsigned int* local_succ, unsigned int* src_ptr, unsigned int* succ_ptr,
unsigned int* K, unsigned int* C,unsigned int iter, float p_t){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;

}