#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"

#define BLOCKS 8
#define TPB 256




__host__ void Check_Out_csv_edge(edge* edge_list, int size){
    ofstream myfile;
    myfile.open(CLUSTER_PATH);
    myfile <<"from,to,cluster\n";
    for(int i=0; i<size;i++){
        myfile<< to_string(edge_list[i].start);
        myfile<< ",";
        myfile<< to_string(edge_list[i].end);
        myfile<< ",";
        myfile<< to_string(edge_list[i].cluster);
        myfile<< "\n";
    }
    myfile.close();
}


__host__ void Check_Out_pref_sum(unsigned long int* list_1, unsigned long int* list_2, int size){
    ofstream myfile;
    myfile.open(LIST_PATH);
    myfile <<"i,List1,List2,List2Check\n";
    unsigned long int* check = new unsigned long int[size];
    check[0]=0;
    for(int i=0; i<size;i++){
        myfile<< to_string(i);
        myfile<< ",";
        if(i>0){
            check[i]=list_1[i-1]+check[i-1];
        }
        myfile<< to_string(list_1[i]);
        myfile<< ",";
        myfile<< to_string(list_2[i]);
        myfile<< ",";
        myfile<< to_string(check[i]);
        myfile<< "\n";
        if(check[i]!=list_2[i]){
            cout<<"Rugh rogh raggy, reheheheheh"<<endl;
        }
    }
    myfile.close();
    delete[] check;
}


__host__ void return_edge_list(string path, edge* arr){
    ifstream data;
    data.open(path);
    string line,word;
    unsigned long int count=0;
    unsigned int column=0;
    cout<<data.is_open()<<endl;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); //Stream Class to operate on strings
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        arr[count-1].start=stoul(word);
                        column++;
                    }
                    else{
                        arr[count-1].end=stoul(word);
                        arr[count-1].cluster=0u;
                    }
                }
                //Extract data until ',' is found
            }
            column=0;
            count++;
        }
    }
    else{
        cout<<"Cannot open file"<<endl;
    }
    cout<<count<<endl;
    data.close();
}

__host__ void get_graph_info(string path, unsigned long int* nodes, unsigned long int* edges){
    ifstream data;
    data.open(path);
    string line,word;
    int count =0;
    int column = 0;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); 
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        cout<<word<<endl;
                        *nodes=stoi(word);
                        column++;
                    }
                    else{
                        *edges=stoi(word);
                    }
                }
                //Extract data until ',' is found
            }
            count++;
        }
    }

}


__host__ void split_list(unsigned int** arr, unsigned int* subarr_1, unsigned int* subarr_2, unsigned int size){
    for(unsigned int i=0; i<size;i++){
        subarr_1[i]=arr[i][0];
        subarr_2[i]=arr[i][1];
    }
}


__global__ void Sort_Cluster(edge* edgelist, unsigned long int* table, unsigned long int size,unsigned int iter){
    //Need to sort through the cluster data and organize it
    //organize into the data for each block of FrogWild
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    __shared__ edge shared_edge[TPB];
    __shared__ unsigned long int bits[TPB];
    __shared__ unsigned long int ex_bits[TPB+1];
    //Load vertex and cluster info into the shared memory
    if(idx<size){
        shared_edge[tid].cluster=edgelist[idx].cluster;
        shared_edge[tid].end=edgelist[idx].end;
        shared_edge[tid].start=edgelist[idx].start;
    }
    __syncthreads();

    //Perform sorting
    unsigned long int key, bit;
    int from, to;
    if(idx<size){
        key = shared_edge[tid].cluster;
        from = shared_edge[tid].start;
        to = shared_edge[tid].end;
        bit=(key>>iter) & 0x0001;
        bits[tid]=bit;
    }
    __syncthreads();
    //Perform exclusive scan
    if(idx<size && tid!=0){
        ex_bits[tid]=bits[tid-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    unsigned long int num_one_total;
    if(idx==size-1 || tid == blockDim.x-1){
        ex_bits[blockDim.x]=bits[tid]+ex_bits[tid];
        table[blockIdx.x]=(idx==size-1)?(size-(blockIdx.x*blockDim.x+ex_bits[blockDim.x])):(TPB-ex_bits[blockDim.x]);
        //Save the number of 1's
        table[blockIdx.x+gridDim.x]=ex_bits[blockDim.x];
    }
    __syncthreads();
    if(idx<size){
        unsigned long int num_one_bef=ex_bits[tid];
        unsigned long int num_one_total=ex_bits[blockDim.x];
        unsigned long int dst = (1-bit)*(tid - num_one_bef)+ bit*(blockDim.x-num_one_total+num_one_bef);
        shared_edge[dst].cluster=key;
        shared_edge[dst].start=from;
        shared_edge[dst].end=to;
    }
    __syncthreads();
    if(idx<size){
        edgelist[idx].cluster=shared_edge[tid].cluster;
        edgelist[idx].start=shared_edge[tid].start;
        edgelist[idx].end=shared_edge[tid].end;
    }
}

__global__ void Swap(edge* edge_list, unsigned long int* table, unsigned long int* table_2, long int size, unsigned int iter){
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    // const unsigned int cluster_size= size/gridDim.x+1;
    __shared__ edge shared_edge[TPB];
    //Load vertex and cluster info into the shared memory
    unsigned int bit, key, dst;
    if(idx<size){
        shared_edge[tid].cluster=edge_list[idx].cluster;
        shared_edge[tid].end=edge_list[idx].end;
        shared_edge[tid].start=edge_list[idx].start;
        key = shared_edge[tid].cluster;
        bit =  (key>>iter) & 1;
    }
    __syncthreads();   
    if(idx<size){
        // dst=(bit==0)?(table_2[blockIdx.x]+tid):(table_2[blockIdx.x+(gridDim.x)]+tid-(table[blockIdx.x]));
        dst = (bit==0)? (tid+table_2[blockIdx.x]):(tid-table[blockIdx.x]+table_2[blockIdx.x+gridDim.x]);
        edge_list[dst].cluster=shared_edge[tid].cluster;
        edge_list[dst].end=shared_edge[tid].end;
        edge_list[dst].start=shared_edge[tid].start;
    }
    __syncthreads();
}

__global__ void bit_exclusive_scan(unsigned long int* bits, unsigned long int* bits_2, unsigned long int* bits_3, unsigned long int size){
    unsigned int tid=threadIdx.x;
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    __shared__ unsigned int ex_bits[TPB];
    if(idx<size && idx!=0){
        ex_bits[tid]=bits[idx-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    if(idx<size){
        bits_2[idx]=ex_bits[tid];
    }
    if(tid==TPB-1){
        bits_3[blockIdx.x]=ex_bits[tid];
    }
    __syncthreads();
}

__global__ void fin_exclusive_scan(unsigned long int* bits_3, unsigned long int size){
    unsigned long int tid = threadIdx.x;
    unsigned long int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    __syncthreads();
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=bits_3[tid]+bits_3[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            bits_3[tid]=temp;
        }
    }
}

__global__ void final_scan_commit(unsigned long int* bits_2, unsigned long int* bits_3, unsigned long int size){
    unsigned int bid = blockIdx.x;
    unsigned int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    if(idx<size && bid>0){
        bits_2[idx]+=bits_3[bid-1];
    }
}


//d_table_2 contains the prefix sum
//d_table contains the counts

__host__ void Org_Vertex_Helper(edge* h_edge, unsigned long int size){
    //Allocate memory for vertex and cluster info
    edge* d_edge;
    unsigned long int* d_table;
    unsigned long int* d_table_2;
    unsigned long int* d_table_3;

    unsigned long int threads_per_block=TPB;
    unsigned long int blocks_per_grid= size/threads_per_block+1;
    cout<<"Num of blocks "<<blocks_per_grid<<endl;
    unsigned long int ex_block_pg=(2*blocks_per_grid)/threads_per_block+1;
    cout<<"Second amount of blocks "<< ex_block_pg <<endl;
    
    unsigned long int* h_table=new unsigned long int[2*blocks_per_grid];
    unsigned long int* h_table_2=new unsigned long int[2*blocks_per_grid];
    if(!HandleCUDAError(hipMalloc((void**) &d_edge, size*sizeof(edge)))){
        cout<<"Unable to allocate memory for vertex data"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**) &d_table,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table,0,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to set table to 0"<<endl;
    }

    if(!HandleCUDAError(hipMalloc((void**) &d_table_2,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table_2,0,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to set table to 0"<<endl;
    }

    if(!HandleCUDAError(hipMalloc((void**) &d_table_3,(ex_block_pg)*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table_3,0,(ex_block_pg)*sizeof(unsigned long int)))){
        cout<<"Unable to set table to 0"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_edge,h_edge,size*sizeof(edge), hipMemcpyHostToDevice))){
        cout<<"Unable to copy cluster data"<<endl;
    }
    double r = ((double) rand() / (RAND_MAX));
    Random_Edge_Placement<<<blocks_per_grid,threads_per_block>>>(d_edge, r);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Rand_Edge Place"<<endl;
    } 
    if(ex_block_pg>0){
        for(unsigned int i=0; i<=(unsigned int)log2(double(BLOCKS));i++){
            Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
            }
            bit_exclusive_scan<<<ex_block_pg,threads_per_block>>>(d_table,d_table_2,d_table_3,2*blocks_per_grid);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host exclusive scan"<<endl;
            }
            fin_exclusive_scan<<<1,ex_block_pg,sizeof(int)*ex_block_pg>>>(d_table_3,ex_block_pg);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host for final exclusive scan"<<endl;
            }
            final_scan_commit<<<ex_block_pg,threads_per_block>>>(d_table_2,d_table_3,2*blocks_per_grid);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host for final exclusive scan commit"<<endl;
            }
            Swap<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,d_table_2,size, i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host swap"<<endl;
            }
        }
    }
    else{
        for(unsigned int i=0; i<32;i++){
            Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
            }
            bit_exclusive_scan<<<ex_block_pg,threads_per_block>>>(d_table,d_table_2,d_table_3,2*blocks_per_grid);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host exclusive scan"<<endl;
            }
            Swap<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,d_table_2,size, i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host swap"<<endl;
            }
        }
    }


    if(!HandleCUDAError(hipMemcpy(h_edge,d_edge,size*sizeof(edge),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back edge data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_table,d_table,2*blocks_per_grid*sizeof(unsigned long int),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back edge data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_table_2,d_table_2,2*blocks_per_grid*sizeof(unsigned long int),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back edge data"<<endl;
    }

    Check_Out_pref_sum(h_table,h_table_2,2*blocks_per_grid);

    delete[] h_table;
    delete[] h_table_2;

    HandleCUDAError(hipFree(d_edge));
    HandleCUDAError(hipFree(d_table));
    HandleCUDAError(hipFree(d_table_2));
    HandleCUDAError(hipFree(d_table_3));
    HandleCUDAError(hipDeviceReset());   
}



__host__ graph *create_graph (edge *edges){
   int i;
   struct graph *graph = (struct graph *) malloc (sizeof (struct graph));
   for (i = 0; i < NODES; i++) {
      graph->point[i] = NULL;
   }
   for (i = 0; i < EDGES; i++) {
      int start = edges[i].start;
      int end = edges[i].end;
      struct vertex *v = (struct vertex *) malloc (sizeof (struct vertex));
      v->end = end;
      v->next = graph->point[start];
      graph->point[start] = v;
   }
   return graph;
}

__global__ void Random_Edge_Placement(edge *edges, double rand_num){
    unsigned int idx= threadIdx.x+blockDim.x*blockIdx.x;
    __syncthreads();
    //Use multiplication hashing
    double intpart;
    double mod_part = modf(idx*rand_num, &intpart);
    unsigned int hash = (unsigned int)(BLOCKS*mod_part);
    //We now have the key, we need to sort
    if(idx<EDGES){
        edges[idx].cluster=hash;
    }
    __syncthreads();

}