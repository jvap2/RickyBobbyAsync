#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"

#define BLOCKS 32
#define TPB 256


// __host__ graph *create_graph (edge *edges){
//    int i;
//    struct graph *graph = (struct graph *) malloc (sizeof (struct graph));
//    for (i = 0; i < NODES; i++) {
//       graph->point[i] = NULL;
//    }
//    for (i = 0; i < EDGES; i++) {
//       int start = edges[i].start;
//       int end = edges[i].end;
//       struct vertex *v = (struct vertex *) malloc (sizeof (struct vertex));
//       v->end = end;
//       v->next = graph->point[start];
//       graph->point[start] = v;
//    }
//    return graph;
// }

__host__ void Check_Out_csv_edge(edge* edge_list, int size){
    ofstream myfile;
    myfile.open(CLUSTER_PATH);
    myfile <<"from,to,cluster\n";
    for(int i=0; i<size;i++){
        myfile<< to_string(edge_list[i].start);
        myfile<< ",";
        myfile<< to_string(edge_list[i].end);
        myfile<< ",";
        myfile<< to_string(edge_list[i].cluster);
        myfile<< "\n";
    }
    myfile.close();
}


__host__ void Check_Out_pref_sum(unsigned long int* list_1, unsigned long int* list_2, int size){
    ofstream myfile;
    myfile.open(LIST_PATH);
    myfile <<"i,List1,List2,List2Check\n";
    unsigned long int* check = new unsigned long int[size];
    check[0]=0;
    for(int i=0; i<size;i++){
        myfile<< to_string(i);
        myfile<< ",";
        if(i>0){
            check[i]=list_1[i-1]+check[i-1];
        }
        myfile<< to_string(list_1[i]);
        myfile<< ",";
        myfile<< to_string(list_2[i]);
        myfile<< ",";
        myfile<< to_string(check[i]);
        myfile<< "\n";
        if(check[i]!=list_2[i]){
            cout<<"Rugh rogh raggy, reheheheheh"<<endl;
        }
    }
    myfile.close();
    delete[] check;
}


__host__ void return_edge_list(string path, edge* arr){
    ifstream data;
    data.open(path);
    string line,word;
    unsigned long int count=0;
    unsigned int column=0;
    cout<<data.is_open()<<endl;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); //Stream Class to operate on strings
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        arr[count-1].start=stoul(word);
                        column++;
                    }
                    else{
                        arr[count-1].end=stoul(word);
                        arr[count-1].cluster=0u;
                    }
                }
                //Extract data until ',' is found
            }
            column=0;
            count++;
        }
    }
    else{
        cout<<"Cannot open file"<<endl;
    }
    cout<<count<<endl;
    data.close();
}

__host__ void get_graph_info(string path, unsigned long int* nodes, unsigned long int* edges){
    ifstream data;
    data.open(path);
    string line,word;
    int count =0;
    int column = 0;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); 
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        cout<<word<<endl;
                        *nodes=stoi(word);
                        column++;
                    }
                    else{
                        *edges=stoi(word);
                    }
                }
                //Extract data until ',' is found
            }
            count++;
        }
    }

}


__host__ void split_list(unsigned int** arr, unsigned int* subarr_1, unsigned int* subarr_2, unsigned int size){
    for(unsigned int i=0; i<size;i++){
        subarr_1[i]=arr[i][0];
        subarr_2[i]=arr[i][1];
    }
}


// C++ implementation of Radix Sort


// A utility function to get maximum
// value in arr[]
__host__ int getMax_cluster(edge* edge_list, int n)
{
    int mx = edge_list[0].cluster;
    for (int i = 1; i < n; i++)
        if (edge_list[i].cluster > mx)
            mx = edge_list[i].cluster;
    return mx;
}

// A function to do counting sort of arr[]
// according to the digit
// represented by exp.
__host__ void cpu_countSort(edge* arr, int n, int exp)
{

    // Output array
    edge* out;
    out=(edge*)malloc(sizeof(edge)*n);
    int i, count[10] = { 0 };

    // Store count of occurrences
    // in count[]
    for (i = 0; i < n; i++)
        count[(arr[i].cluster / exp) % 10]++;

    // Change count[i] so that count[i]
    // now contains actual position
    // of this digit in output[]
    for (i = 1; i < 10; i++)
        count[i] += count[i - 1];

    // Build the output array
    for (i = n - 1; i >= 0; i--) {
        out[count[(arr[i].cluster / exp) % 10] - 1] = arr[i];
        count[(arr[i].cluster / exp) % 10]--;
    }

    // Copy the output array to arr[],
    // so that arr[] now contains sorted
    // numbers according to current digit
    for (i = 0; i < n; i++)
        arr[i] = out[i];

    free(out);
}

// The main function to that sorts arr[]
// of size n using Radix Sort
__host__ void cpu_radixsort(edge* arr, int n)
{

    // Find the maximum number to
    // know number of digits
    int m = getMax_cluster(arr, n);

    // Do counting sort for every digit.
    // Note that instead of passing digit
    // number, exp is passed. exp is 10^i
    // where i is current digit number
    for (int exp = 1; m / exp > 0; exp *= 10)
        cpu_countSort(arr, n, exp);
}


__host__ void Org_Vertex_Helper(edge* h_edge, unsigned long int size){
    //Allocate memory for vertex and cluster info
    edge* d_edge;
    edge* d_edge_2;
    unsigned long int* d_table;
    unsigned long int* d_table_2;
    unsigned long int* d_table_3;

    unsigned long int threads_per_block=TPB;
    unsigned long int blocks_per_grid= size/threads_per_block+1;
    cout<<"Num of blocks "<<blocks_per_grid<<endl;
    unsigned long int ex_block_pg=(2*blocks_per_grid)/threads_per_block+1;
    cout<<"Second amount of blocks "<< ex_block_pg <<endl;
    
    if(!HandleCUDAError(hipMalloc((void**) &d_edge, size*sizeof(edge)))){
        cout<<"Unable to allocate memory for vertex data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_edge,h_edge,size*sizeof(edge), hipMemcpyHostToDevice))){
        cout<<"Unable to copy cluster data"<<endl;
    }

    unsigned long int* d_hist;
    unsigned long int* dev_fin_hist;
    unsigned long int* dev_fin_count;
    // unsigned int* h_hist= new unsigned int [BLOCKS*blocks_per_grid];


    if(!HandleCUDAError(hipMalloc((void**)&d_hist, BLOCKS*blocks_per_grid*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&dev_fin_hist, BLOCKS*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&dev_fin_count, BLOCKS*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    double r = ((double) rand() / (RAND_MAX));
    Random_Edge_Placement<<<blocks_per_grid,threads_per_block>>>(d_edge, r);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Rand_Edge Place"<<endl;
    }
    Histogram_1<<<blocks_per_grid,threads_per_block>>>(d_edge,d_hist,size); 
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Hist_1"<<endl;
    }
    Kogge_Stone_Hist_Reduct<<<BLOCKS,blocks_per_grid, blocks_per_grid*sizeof(unsigned long int)>>>(d_hist,dev_fin_hist,BLOCKS*blocks_per_grid);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host for reduce"<<endl;
    }
    Hist_Prefix_Sum<<<1,BLOCKS>>>(dev_fin_hist, dev_fin_count);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host for reduce"<<endl;
    }
    HandleCUDAError(hipFree(d_hist));
    if(!HandleCUDAError(hipMalloc((void**) &d_edge_2, size*sizeof(edge)))){
        cout<<"Unable to allocate memory for vertex data"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**) &d_table,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table,0,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to set table to 0"<<endl;
    }

    if(!HandleCUDAError(hipMalloc((void**) &d_table_2,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table_2,0,(2*blocks_per_grid)*sizeof(unsigned long int)))){
        cout<<"Unable to set table to 0"<<endl;
    }

    if(!HandleCUDAError(hipMalloc((void**) &d_table_3,(ex_block_pg)*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for the table data"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_table_3,0,(ex_block_pg)*sizeof(unsigned long int)))){
        cout<<"Unable to set table to 0"<<endl;
    }
    if(ex_block_pg>0){
        for(unsigned int i=0; i<=(unsigned int)log2(double(BLOCKS));i++){
            Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
            }
            bit_exclusive_scan<<<ex_block_pg,threads_per_block>>>(d_table,d_table_2,d_table_3,2*blocks_per_grid);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host exclusive scan"<<endl;
            }
            fin_exclusive_scan<<<1,ex_block_pg,sizeof(int)*ex_block_pg>>>(d_table_3,ex_block_pg);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host for final exclusive scan"<<endl;
            }
            final_scan_commit<<<ex_block_pg,threads_per_block>>>(d_table_2,d_table_3,2*blocks_per_grid);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host for final exclusive scan commit"<<endl;
            }
            Swap<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2,d_table, d_table_2,size, i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host swap"<<endl;
            }
            copy_edge_list<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2,size);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host swap"<<endl;
            }
        }
    }
    else{
        for(unsigned int i=0; i<32;i++){
            Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
            }
            bit_exclusive_scan<<<ex_block_pg,threads_per_block>>>(d_table,d_table_2,d_table_3,2*blocks_per_grid);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host exclusive scan"<<endl;
            }
            Swap<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2, d_table,d_table_2,size, i);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host swap"<<endl;
            }
            copy_edge_list<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2,size);
            if(!HandleCUDAError(hipDeviceSynchronize())){
                cout<<"Unable to synchronize with host swap"<<endl;
            }
        }
    }
    HandleCUDAError(hipFree(d_edge_2));
    HandleCUDAError(hipFree(d_table));
    HandleCUDAError(hipFree(d_table_2));
    HandleCUDAError(hipFree(d_table_3));
    HandleCUDAError(hipFree(dev_fin_hist));

    if(!HandleCUDAError(hipMemcpy(h_edge,d_edge,size*sizeof(edge),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back edge data"<<endl;
    }
    unsigned long int *d_K, *d_c;
    if(!HandleCUDAError(hipMalloc((void**)&d_K, size*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for K"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_c, size*sizeof(unsigned long int)))){
        cout<<"Unable to allocate memory for c"<<endl;
    }

    HandleCUDAError(hipFree(d_edge));
    HandleCUDAError(hipDeviceReset());   
}

__global__ void Sort_Cluster(edge* edgelist, unsigned long int* table, unsigned long int size,unsigned int iter){
    //Need to sort through the cluster data and organize it
    //organize into the data for each block of FrogWild
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    __shared__ edge shared_edge[TPB];
    __shared__ unsigned long int bits[TPB];
    __shared__ unsigned long int ex_bits[TPB+1];
    //Load vertex and cluster info into the shared memory
    if(idx<size){
        shared_edge[tid]=edgelist[idx];
    }
    __syncthreads();

    //Perform sorting
    unsigned long int key, bit;
    int from, to;
    if(idx<size){
        key = shared_edge[tid].cluster;
        from = shared_edge[tid].start;
        to = shared_edge[tid].end;
        bit=(key>>iter) & 1;
        bits[tid]=bit;
    }
    __syncthreads();
    //Perform exclusive scan
    if(idx<size && tid!=0){
        ex_bits[tid]=bits[tid-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    unsigned long int num_one_total;
    if(idx==size-1 || tid == blockDim.x-1){
        ex_bits[blockDim.x]=bits[tid]+ex_bits[tid];
        table[blockIdx.x]=(idx==size-1)?(size-(blockIdx.x*blockDim.x+ex_bits[blockDim.x])):(TPB-ex_bits[blockDim.x]);
        //Save the number of 1's
        table[blockIdx.x+gridDim.x]=ex_bits[blockDim.x];
    }
    __syncthreads();
    if(idx<size){
        unsigned long int num_one_bef=ex_bits[tid];
        unsigned long int num_one_total=ex_bits[blockDim.x];
        unsigned long int dst = (1-bit)*(tid - num_one_bef)+ bit*(blockDim.x-num_one_total+num_one_bef);
        shared_edge[dst].cluster=key;
        shared_edge[dst].start=from;
        shared_edge[dst].end=to;
    }
    __syncthreads();
    if(idx<size){
        edgelist[idx]=shared_edge[tid];
        //The edge list is now sorted block-wise
    }
}

__global__ void Swap(edge* edge_list, edge* edge_list_2, unsigned long int* table, unsigned long int* table_2, long int size, unsigned int iter){
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    // const unsigned int cluster_size= size/gridDim.x+1;
    __shared__ edge shared_edge[TPB];
    //Load vertex and cluster info into the shared memory
    unsigned int bit, key, dst;
    if(idx<size){
        shared_edge[tid]=edge_list[idx];
        key = shared_edge[tid].cluster;
        bit =  (key>>iter) & 1;
    }
    __syncthreads();   
    if(idx<size){
        dst = (bit==0)? (tid+table_2[blockIdx.x]):(tid-table[blockIdx.x]+table_2[blockIdx.x+gridDim.x]);
        edge_list_2[dst]=shared_edge[tid];
    }
}

__global__ void bit_exclusive_scan(unsigned long int* bits, unsigned long int* bits_2, unsigned long int* bits_3, unsigned long int size){
    unsigned int tid=threadIdx.x;
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    __shared__ unsigned int ex_bits[TPB];
    if(idx<size && idx!=0){
        ex_bits[tid]=bits[idx-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    if(idx<size){
        bits_2[idx]=ex_bits[tid];
    }
    if(tid==TPB-1){
        bits_3[blockIdx.x]=ex_bits[tid];
    }
    __syncthreads();
}

__global__ void fin_exclusive_scan(unsigned long int* bits_3, unsigned long int size){
    unsigned long int tid = threadIdx.x;
    unsigned long int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    __syncthreads();
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=bits_3[tid]+bits_3[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            bits_3[tid]=temp;
        }
    }
}

__global__ void final_scan_commit(unsigned long int* bits_2, unsigned long int* bits_3, unsigned long int size){
    unsigned int bid = blockIdx.x;
    unsigned int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    if(idx<size && bid>0){
        bits_2[idx]+=bits_3[bid-1];
    }
}


//d_table_2 contains the prefix sum
//d_table contains the counts
__global__ void copy_edge_list(edge* edge_1, edge* edge_2, unsigned long int size){
    unsigned int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        edge_1[idx]=edge_2[idx];
    }
}


__global__ void Random_Edge_Placement(edge *edges, double rand_num){
    unsigned int idx= threadIdx.x+blockDim.x*blockIdx.x;
    __syncthreads();
    //Use multiplication hashing
    double intpart;
    double mod_part = modf(idx*rand_num, &intpart);
    unsigned long int hash = (unsigned int)(BLOCKS*mod_part);
    //We now have the key, we need to sort
    if(idx<EDGES){
        edges[idx].cluster=hash;
    }
    __syncthreads();

}





__global__ void Histogram_1(edge* edgelist, unsigned long int* hist_bin, unsigned long int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ unsigned int s_edge_list[TPB];
    __shared__ unsigned int s_hist[BLOCKS];
    if(idx<size){
        s_edge_list[tid]=edgelist[idx].cluster;
        //Copy TPB cluster values over
    }
    __syncthreads();
    if(idx<size){
        atomicAdd(&s_hist[s_edge_list[tid]],1);
    }
    __syncthreads();
    if(tid<BLOCKS){
        hist_bin[BLOCKS*blockIdx.x+tid]=s_hist[tid];
    }
    //Now, all the data is stored locally on a blocks/grid by BLOCKS array which we need to reduce
}

__global__ void Kogge_Stone_Hist_Reduct(unsigned long int* hist_bin, unsigned long int* fin_bin, int size){
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ unsigned long int clust_val[];
    if(idx<size){
        clust_val[tid]=hist_bin[tid*BLOCKS+blockIdx.x];
    }
    else{
        clust_val[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=clust_val[tid]+clust_val[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            clust_val[tid]=temp;
        }
    }
    if(tid==blockDim.x){
        fin_bin[blockIdx.x]=clust_val[tid];
    }
    __syncthreads();
}

__global__ void Hist_Prefix_Sum(unsigned long int* fin_bin, unsigned long int* fin_bin_2){
    unsigned int tid = threadIdx.x+blockDim.x*blockIdx.x;
    __shared__ unsigned long int local[BLOCKS];
    if(tid<BLOCKS && tid!=0){
        local[tid]=fin_bin[tid-1];
    }
    else{
        local[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned long int temp;
        if(tid>=stride){
            temp=local[tid]+local[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            local[tid]=temp;
        }
    }
    if(tid<BLOCKS){
        fin_bin_2[tid]=local[tid];
    }
}

__global__ void Build_Partition_Vertices(edge* edgelist, vertex* vert_list, unsigned long int* ptr_list, unsigned long int* ctr_list, int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    edge* local_edge=edgelist+ptr_list[blockIdx.x];
    extern __shared__ edge shared_edge[];
    extern __shared__ unsigned int src[];

    if(idx<size){
        for(int i=tid; i<ctr_list[blockIdx.x+1];i+=blockDim.x){
            shared_edge[i]=local_edge[i];
        }
    }
    __syncthreads();
    //Lines below assume that the list has been sorted already
    if(idx<size){
        for(int i=tid; i<ctr_list[blockIdx.x+1];i+=blockDim.x){
            src[i]=(int)(shared_edge[i].start<shared_edge[i+1].end);//1 if different, 0 if the same
        }
    }


}