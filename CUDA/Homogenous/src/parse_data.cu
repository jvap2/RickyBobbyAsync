#include "hip/hip_runtime.h"
#include "../include/data.h"
#include "../include/GPUErrors.h"





__host__ void Check_Out_csv_edge(edge* edge_list, int size){
    ofstream myfile;
    myfile.open(CLUSTER_PATH);
    myfile <<"from,to,cluster\n";
    for(int i=0; i<size;i++){
        myfile<< to_string(edge_list[i].start);
        myfile<< ",";
        myfile<< to_string(edge_list[i].end);
        myfile<< ",";
        myfile<< to_string(edge_list[i].cluster);
        myfile<< "\n";
    }
    myfile.close();
}

__host__ void Check_Out_Ptr_Ctr(unsigned int* h_ctr, unsigned int* h_ptr, int size){
    ofstream myfile;
    myfile.open(PTR_CTR_PATH);
    myfile <<"ptr,ctr\n";
    for(int i=0; i<size;i++){
        myfile<< to_string(h_ptr[i]);
        myfile<< ",";
        myfile<< to_string(h_ctr[i]);
        myfile<< "\n";
    }
    myfile.close();
}

__host__ void Check_Out_Unq(unsigned int* h_unq, int size){
    ofstream myfile;
    myfile.open(UNQ_PATH);
    myfile <<"unq\n";
    for(int i=0; i<size;i++){
        myfile<< to_string(h_unq[i]);
        myfile<< "\n";
    }
    myfile.close();
}

__host__ void Check_Out_ptr(unsigned int* edge_list, int size){
    ofstream myfile;
    myfile.open(PTR_PATH);
    myfile <<"ptr\n";
    for(int i=0; i<size;i++){
        myfile<< to_string(edge_list[i]);
        myfile<< "\n";
    }
    myfile.close();
}


__host__ void Check_Out_pref_sum(unsigned int* list_1, unsigned int* list_2, int size){
    ofstream myfile;
    myfile.open(LIST_PATH);
    myfile <<"i,List1,List2,List2Check\n";
    unsigned int* check = new unsigned int[size];
    check[0]=0;
    for(int i=0; i<size;i++){
        myfile<< to_string(i);
        myfile<< ",";
        if(i>0){
            check[i]=list_1[i-1]+check[i-1];
        }
        myfile<< to_string(list_1[i]);
        myfile<< ",";
        myfile<< to_string(list_2[i]);
        myfile<< ",";
        myfile<< to_string(check[i]);
        myfile<< "\n";
        if(check[i]!=list_2[i]){
            cout<<"Rugh rogh raggy, reheheheheh"<<endl;
        }
    }
    myfile.close();
    delete[] check;
}


__host__ void check_out_replicas(string path,unsigned int* replicas, unsigned int node_size){
    unsigned int total_rep;
    float rep_avg;
    total_rep=0;
    rep_avg=0;
    for(int i=0; i<node_size;i++){
        total_rep+=replicas[i];
    }
    rep_avg=1.0f*total_rep/(1.0f*node_size);
    ofstream myfile;
    myfile.open(path, ios::app);
    myfile<< to_string(node_size);
    myfile<< ",";
    myfile<< to_string(total_rep);
    myfile<< ",";
    myfile<< to_string(rep_avg);
    myfile<< ",";
    myfile<< to_string(BLOCKS);
    myfile<< "\n";
    myfile.close();
}


__host__ void return_edge_list(string path, edge* arr){
    cout<<"Getting edge list"<<endl;
    ifstream data;
    data.open(path);
    string line,word;
    unsigned int count=0;
    unsigned int column=0;
    cout<<"Data is open "<<data.is_open()<<endl;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); //Stream Class to operate on strings
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        arr[count-1].start=stoul(word);
                        column++;
                    }
                    else{
                        arr[count-1].end=stoul(word);
                        arr[count-1].cluster=0u;
                    }
                }
                //Extract data until ',' is found
            }
            column=0;
            count++;
        }
    }
    else{
        cout<<"Cannot open file"<<endl;
    }
    cout<<count<<endl;
    data.close();
}

__host__ void Check_Repeats(edge* edge_list, unsigned int size){
    for(int i=1; i<size;i++){
        if(edge_list[i].start==edge_list[i-1].start && edge_list[i].end==edge_list[i-1].end && edge_list[i].cluster==edge_list[i-1].cluster){
            cout<<"Repeat at "<<i<<endl;
        }
    }
}

__host__ void CSR_Graph(string path, unsigned int node_size, unsigned int edge_size, unsigned int* src_ptr, unsigned int* succ, unsigned int* deg_arr){
    ifstream data;
    data.open(path);
    string line,word;
    unsigned int count = 0;
    unsigned int column=0;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); //Stream Class to operate on strings
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        src_ptr[stoi(word)]++; //Create a histogram of values
                        deg_arr[stoi(word)]++;
                        column++;
                    }
                    else{
                        succ[count-1]=stoul(word);

                    }
                }
                //Extract data until ',' is found
            }
            column=0;
            count++;
        }
    }
    else{
        cout<<"Cannot open file"<<endl;
    }
    //Perform prefix sum of src_prt
    unsigned int* copy_ptr = new unsigned int[node_size+1];
    copy_ptr[0]=0;
    for(unsigned int i=1; i<node_size+1;i++){
        copy_ptr[i]=src_ptr[i-1];
    }
    src_ptr[0]=0;
    for(unsigned int i=1; i<node_size+1;i++){
        copy_ptr[i]+=copy_ptr[i-1];
        src_ptr[i]=copy_ptr[i];
    }
    cout<<count<<endl;
    data.close();
}

__host__ void Capture_Node_Degree(edge* edge_list, unsigned int* deg_arr, unsigned int size){
    for(unsigned int i=0; i<size;i++){
        deg_arr[edge_list[i].start]++;
    }
}

__host__ void get_graph_info(string path, unsigned int* nodes, unsigned int* edges){
    cout<<"Getting graph info"<<endl;
    ifstream data;
    data.open(path);
    string line,word;
    int count =0;
    int column = 0;
    if(data.is_open()){
        //Check if data is open
        while(getline(data,line)){
            //Keep extracting data until a delimiter is found
            stringstream stream_data(line); 
            while(getline(stream_data,word,',')){
                if(count==0){
                    continue;
                }
                else{
                    if(column==0){
                        cout<<word<<endl;
                        *nodes=stoi(word);
                        column++;
                    }
                    else{
                        *edges=stoi(word);
                    }
                }
                //Extract data until ',' is found
            }
            count++;
        }
    }

}


__host__ void Export_Local_Src(unsigned int* local_src, unsigned int* h_ptr, unsigned int* h_ctr){
    ofstream myfile;
    myfile.open(LOCAL_SRC_PATH);
    myfile <<"cluster,src\n";
    for(int i = 0; i<BLOCKS; i++){
        for(int j = h_ptr[i]; j<h_ptr[i]+h_ctr[i];j++){
            myfile<< to_string(i);
            myfile<< ",";
            myfile<< to_string(local_src[j]);
            myfile<< "\n";
        }
    }
}

__host__ void Export_Local_Succ(unsigned int* local_succ, unsigned int* h_ptr, unsigned int* h_ctr){
    ofstream myfile;
    myfile.open(LOCAL_SUCC_PATH);
    myfile<<"cluster,succ\n";
    for(int i = 0 ; i< BLOCKS; i++){
        for(int j = h_ptr[i]; j<h_ptr[i]+h_ctr[i];j++){
            myfile<< to_string(i);
            myfile<< ",";
            myfile<< to_string(local_succ[j]);
            myfile<< "\n";
        }
    }
    myfile.close();
}

__host__ void Export_Unq(unsigned int* unq, unsigned int* h_unq_ptr, unsigned int* h_unq_ctr){
    ofstream myfile;
    myfile.open(UNQ_PATH);
    myfile<<"cluster,unq\n";
    for(int i = 0; i<BLOCKS; i++){
        for(int j = h_unq_ptr[i]; j<h_unq_ptr[i]+h_unq_ctr[i];j++){
            myfile<< to_string(i);
            myfile<< ",";
            myfile<< to_string(unq[j]);
            myfile<< "\n";
        }
    }
    myfile.close();
}

__host__ void Export_Unq_Ctr_Ptr(unsigned int* h_unq_ptr, unsigned int* h_unq_ctr){
    ofstream myfile;
    myfile.open(UNQ_CTR_PTR_PATH);
    myfile<<"cluster,unq_ctr,unq_ptr\n";
    for(int i = 0; i<BLOCKS; i++){
        myfile<< to_string(i);
        myfile<< ",";
        myfile<< to_string(h_unq_ctr[i]);
        myfile<< ",";
        myfile<< to_string(h_unq_ptr[i]);
        myfile<< "\n";
    }
    myfile<< to_string(BLOCKS);
    myfile<< ",";
    myfile<< to_string(0);
    myfile<< ",";
    myfile<< to_string(h_unq_ptr[BLOCKS]);
    myfile.close();
}

__host__ void Export_Src_Ctr_Ptr(unsigned int* src_ptr, unsigned int* src_ctr){
    ofstream myfile;
    myfile.open(SRC_CTR_PTR_PATH);
    myfile<<"cluster,src_ctr,src_ptr\n";
    for(int i=0; i<BLOCKS; i++){
        myfile<< to_string(i);
        myfile<< ",";
        myfile<< to_string(src_ctr[i]);
        myfile<< ",";
        myfile<< to_string(src_ptr[i]);
        myfile<< "\n";
    }
    myfile.close();
}

__host__ void Export_H_Ctr_Ptr(unsigned int* h_ptr, unsigned int* h_ctr){
    ofstream myfile;
    myfile.open(H_CTR_PTR_PATH);
    myfile<<"cluster,h_ctr,h_ptr\n";
    for(int i=0; i<BLOCKS;i++){
        myfile<< to_string(i);
        myfile<< ",";
        myfile<< to_string(h_ctr[i]);
        myfile<< ",";
        myfile<< to_string(h_ptr[i]);
        myfile<< "\n";
    }

}

__host__ void Export_Degree(unsigned int* deg, unsigned int node_size){
    ofstream myfile;
    myfile.open(DEG_PATH);
    myfile<<"node,deg\n";
    for(int i=0; i<node_size;i++){
        myfile<< to_string(i);
        myfile<<",";
        myfile<< to_string(deg[i]);
        myfile<<"\n";
    }
    myfile.close();
}


__host__ void Export_Replica_Stats(replica_tracker* h_replica, unsigned int node_size){
    ofstream myfile;
    myfile.open(REPLICA_STAT_PATH);
    myfile<<"node,num_replicas,";
    for(int i=0; i<BLOCKS;i++){
        myfile<<to_string(i);
        myfile<<",";
    }
    myfile<<"\n";
    for(int i=0; i<node_size;i++){
        myfile<< to_string(i);
        myfile<<",";
        myfile<< to_string(h_replica[i].num_replicas);
        myfile<<",";
        for(int j=0; j<BLOCKS;j++){
            myfile<< to_string(h_replica[i].clusters[j]);
            myfile<<",";
        }
        myfile<<"\n";
    }
    myfile.close();
}

__host__ void split_list(unsigned int** arr, unsigned int* subarr_1, unsigned int* subarr_2, unsigned int size){
    for(unsigned int i=0; i<size;i++){
        subarr_1[i]=arr[i][0];
        subarr_2[i]=arr[i][1];
    }
}


// C++ implementation of Radix Sort


// A utility function to get maximum
// value in arr[]
__host__ int getMax_cluster(edge* edge_list, int n)
{
    int mx = edge_list[0].cluster;
    for (int i = 1; i < n; i++)
        if (edge_list[i].cluster > mx)
            mx = edge_list[i].cluster;
    return mx;
}

// A function to do counting sort of arr[]
// according to the digit
// represented by exp.
__host__ void cpu_countSort(edge* arr, int n, int exp)
{

    // Output array
    edge* out;
    out=(edge*)malloc(sizeof(edge)*n);
    int i, count[10] = { 0 };

    // Store count of occurrences
    // in count[]
    for (i = 0; i < n; i++)
        count[(arr[i].cluster / exp) % 10]++;

    // Change count[i] so that count[i]
    // now contains actual position
    // of this digit in output[]
    for (i = 1; i < 10; i++)
        count[i] += count[i - 1];

    // Build the output array
    for (i = n - 1; i >= 0; i--) {
        out[count[(arr[i].cluster / exp) % 10] - 1] = arr[i];
        count[(arr[i].cluster / exp) % 10]--;
    }

    // Copy the output array to arr[],
    // so that arr[] now contains sorted
    // numbers according to current digit
    for (i = 0; i < n; i++)
        arr[i] = out[i];

    free(out);
}

// The main function to that sorts arr[]
// of size n using Radix Sort
__host__ void cpu_radixsort(edge* arr, int n)
{

    // Find the maximum number to
    // know number of digits
    int m = getMax_cluster(arr, n);

    // Do counting sort for every digit.
    // Note that instead of passing digit
    // number, exp is passed. exp is 10^i
    // where i is current digit number
    for (int exp = 1; m / exp > 0; exp *= 10)
        cpu_countSort(arr, n, exp);
}


__host__ void Gen_Local_Src(edge* edge_list, unsigned int* src_ptr,unsigned int* temp_src, unsigned int* unq, unsigned int* h_unq_ctr, unsigned int* h_unq_ptr,
unsigned int* h_ctr, unsigned int* h_ptr){
    for(int i = 0; i<BLOCKS; i++){
        //Point to the start of the edge list
        //iterate through the starts
        for(int j=0; j<h_ctr[i];j++){
            unsigned int start = edge_list[h_ptr[i]+j].start;
            src_ptr[h_unq_ptr[i]+start]++;
        }
    }
    //Now, we need to prefix sum the src_ptr
    for(int i=0; i<BLOCKS; i++){
        temp_src[h_unq_ptr[i]]=0;
        for(int j=h_unq_ptr[i]+1; j<h_unq_ptr[i]+h_unq_ctr[i];j++){
            temp_src[j]=src_ptr[j-1]+temp_src[j-1];
        }
    }
    //Now, we need to copy the data back to src_ptr
    for(int i=0; i<BLOCKS; i++){
        for(int j=h_unq_ptr[i]; j<h_unq_ptr[i]+h_unq_ctr[i];j++){
            src_ptr[j]=temp_src[j];
        }
    }
}

__host__ void Generate_Local_Succ(edge* edgelist, unsigned int* local_src, unsigned int* local_succ, unsigned int* h_unq_ctr, unsigned int* h_unq_ptr, unsigned int* h_ptr){
    for(int i = 0; i<BLOCKS; i++){
        //Point to the start of the edge list
        //iterate through the starts
        for(int j=0; j<h_unq_ctr[i]-1;j++){
            for(int k=local_src[h_unq_ptr[i]+j]+h_ptr[i]; k<local_src[h_unq_ptr[i]+j+1]+h_ptr[i];k++){
                local_succ[k]=edgelist[k].end;
            }
        }
    }
}


__host__ void Generate_Renum_Edgelists(edge* edge_list, edge* edge_list_2, unsigned int* unq, unsigned int* h_ptr, unsigned int* h_ctr, unsigned int* h_unq_ctr, unsigned int* h_unq_ptr){
    for(int i = 0; i<BLOCKS; i++){
        //Point to the start of the edge list
        //iterate through the starts
        for(int j=0; j<h_ctr[i];j++){
            unsigned int start = edge_list[h_ptr[i]+j].start;
            unsigned int end = edge_list[h_ptr[i]+j].end;
            int start_idx = find(unq+h_unq_ptr[i], unq+h_unq_ptr[i]+h_unq_ctr[i], start)-(unq+h_unq_ptr[i]);
            int end_idx = find(unq+h_unq_ptr[i], unq+h_unq_ptr[i]+h_unq_ctr[i], end)-(unq+h_unq_ptr[i]);
            if(start_idx>=h_unq_ctr[i] || end_idx>=h_unq_ctr[i]){
                cout<<"Error: "<<start_idx<<", "<<end_idx<<", "<<h_unq_ctr[i]<<endl;
                return;
            }
            else{
                edge_list_2[h_ptr[i]+j].start=start_idx;
                edge_list_2[h_ptr[i]+j].end=end_idx;
                edge_list_2[h_ptr[i]+j].cluster=edge_list[h_ptr[i]+j].cluster;
            }
        }
    }
}

__global__ void Sort_Cluster(edge* edgelist, unsigned int* table, unsigned int size,unsigned int iter){
    //Need to sort through the cluster data and organize it
    //organize into the data for each block of FrogWild
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    __shared__ edge shared_edge[TPB];
    __shared__ unsigned int bits[TPB];
    __shared__ unsigned int ex_bits[TPB+1];
    //Load vertex and cluster info into the shared memory
    if(idx<size){
        shared_edge[tid].start=edgelist[idx].start;
        shared_edge[tid].end=edgelist[idx].end;
        shared_edge[tid].cluster=edgelist[idx].cluster;
    }
    __syncthreads();

    //Perform sorting
    unsigned int key, bit;
    int from, to;
    if(idx<size){
        key = shared_edge[tid].cluster;
        from = shared_edge[tid].start;
        to = shared_edge[tid].end;
        bit=(key>>iter) & 1;
        bits[tid]=bit;
    }
    __syncthreads();
    //Perform exclusive scan
    if(idx<size && tid!=0){
        ex_bits[tid]=bits[tid-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    unsigned int num_one_total;
    if(idx==size-1 || tid == blockDim.x-1){
        ex_bits[blockDim.x]=bits[tid]+ex_bits[tid];
        //Save the number of 0's
        table[blockIdx.x]=(idx==size-1)?(size-(blockIdx.x*blockDim.x+ex_bits[blockDim.x])):(TPB-ex_bits[blockDim.x]);
        //Save the number of 1's
        table[blockIdx.x+gridDim.x]=ex_bits[blockDim.x-1];
    }
    __syncthreads();
    if(idx<size){
        unsigned int num_one_bef=ex_bits[tid];
        unsigned int num_one_total=ex_bits[blockDim.x];
        unsigned int dst =(bit==0)?(tid-num_one_bef):(blockDim.x-num_one_total+num_one_bef);
        // unsigned int dst = (1-bit)*(tid - num_one_bef)+ bit*(blockDim.x-num_one_total+num_one_bef);
        shared_edge[dst].cluster=key;
        shared_edge[dst].start=from;
        shared_edge[dst].end=to;
    }
    __syncthreads();
    if(idx<size){
        edgelist[idx].start=shared_edge[tid].start;
        edgelist[idx].end=shared_edge[tid].end;
        edgelist[idx].cluster=shared_edge[tid].cluster;
        //The edge list is now sorted block-wise
    }
}

__global__ void Swap(edge* edge_list, edge* edge_list_2, unsigned int* table, unsigned int* table_2, long int size, unsigned int iter){
    unsigned int idx= threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid= threadIdx.x;
    // const unsigned int cluster_size= size/gridDim.x+1;
    __shared__ edge shared_edge[TPB];
    //Load vertex and cluster info into the shared memory
    unsigned int bit, key, dst;
    if(idx<size){
        shared_edge[tid].start=edge_list[idx].start;
        shared_edge[tid].end=edge_list[idx].end;
        shared_edge[tid].cluster=edge_list[idx].cluster;
        if(!edge_list[idx].start && !edge_list[idx].end){
            printf("Swap Error EDGELIST1: %d, %d, %d, %d, %d\n", tid, blockIdx.x, shared_edge[tid].start, shared_edge[tid].end, idx);
        }
        key = shared_edge[tid].cluster;
        bit =  (key>>iter) & 1;
    }
    __syncthreads();   
    if(idx<size){
        dst = (bit==0)? (tid+table_2[blockIdx.x]):(tid-table[blockIdx.x]+table_2[blockIdx.x+gridDim.x]);
        edge_list_2[dst].start=shared_edge[tid].start;
        edge_list_2[dst].end=shared_edge[tid].end;
        edge_list_2[dst].cluster=shared_edge[tid].cluster;
    }
    __syncthreads();
    //Check
    if(idx<size){
        if(!edge_list_2[idx].start && !edge_list_2[idx].end){
            printf("Swap Error EDGELIST2: %d, %d, %d, %d, %d\n", tid, blockIdx.x, edge_list_2[idx].start, edge_list_2[idx].end, idx);
        }
    }
}

__global__ void bit_exclusive_scan(unsigned int* bits, unsigned int* bits_2, unsigned int* bits_3, unsigned int size){
    unsigned int tid=threadIdx.x;
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    __shared__ unsigned int ex_bits[TPB];
    if(idx<size && idx!=0){
        ex_bits[tid]=bits[idx-1];
    }
    else{
        ex_bits[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=ex_bits[tid]+ex_bits[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            ex_bits[tid]=temp;
        }
    }
    if(idx<size){
        bits_2[idx]=ex_bits[tid];
    }
    if(tid==TPB-1){
        bits_3[blockIdx.x]=ex_bits[tid];
    }
    __syncthreads();
}

__global__ void fin_exclusive_scan(unsigned int* bits_3, unsigned int size){
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    __syncthreads();
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=bits_3[tid]+bits_3[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            bits_3[tid]=temp;
        }
    }
}

__global__ void final_scan_commit(unsigned int* bits_2, unsigned int* bits_3, unsigned int size){
    unsigned int bid = blockIdx.x;
    unsigned int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    if(idx<size && bid>0){
        bits_2[idx]+=bits_3[bid-1];
    }
}

__global__ void final_scan_commit_scan(unsigned int* list, unsigned int* end_vals, unsigned int ptr, unsigned int size){
    unsigned int bid = blockIdx.x+ptr;
    unsigned int idx = threadIdx.x + (blockIdx.x*blockDim.x);
    if(idx<size && bid>0){
        list[idx]+=end_vals[bid-1];
    }
}


//d_table_2 contains the prefix sum
//d_table contains the counts
__global__ void copy_edge_list(edge* edge_1, edge* edge_2, unsigned int size){
    unsigned int idx=threadIdx.x+(blockDim.x*blockIdx.x);
    if(idx<size){
        edge_1[idx].start=edge_2[idx].start;
        edge_1[idx].end=edge_2[idx].end;
        edge_1[idx].cluster=edge_2[idx].cluster;
        if(!edge_1[idx].start && !edge_1[idx].end){
            printf("Copying Error: %d, %d, %d, %d, %d\n", idx, threadIdx.x, blockIdx.x, edge_1[idx].start, edge_1[idx].end);
        }
    }
}


__global__ void Random_Edge_Placement(edge *edges, double rand_num, unsigned int size){
    unsigned int idx= threadIdx.x+blockDim.x*blockIdx.x;
    __syncthreads();
    //Use multiplication hashing
    double intpart;
    double mod_part = modf(idx*rand_num, &intpart);
    unsigned int hash = (unsigned int)(BLOCKS*mod_part);
    //We now have the key, we need to sort
    if(idx<size){
        edges[idx].cluster=hash;
    }
    __syncthreads();

}


/*CHECK THIS ONE- MAKE SURE THE CSR FORMAT IS PROPER*/
__global__ void Degree_Based_Placement(edge* edges, unsigned int* deg_arr, double rand_num, replica_tracker* d_rep, unsigned int size){
    unsigned int idx= threadIdx.x+blockDim.x*blockIdx.x;
    if(idx<size){
        unsigned int start = edges[idx].start;
        unsigned int end = edges[idx].end;
        unsigned int deg_start = deg_arr[start];
        unsigned int deg_end = deg_arr[end];
        unsigned int v_hash = (deg_start>deg_end)?start:end;
        double intpart;
        double mod_part = modf(v_hash*rand_num, &intpart);
        unsigned int hash = (unsigned int)floor(BLOCKS*mod_part);
        // int hash = v_hash%BLOCKS;
        edges[idx].cluster=hash;
        //Now, we need to update the replica tracker
        /*We are going to need to use some atomic form to be able to write correctly*/
        atomicOr(&d_rep[start].clusters[hash],1);
        atomicOr(&d_rep[end].clusters[hash],1);

    }

}


/*We will now need to reduce the d_rep stuff*/

__global__ void Finalize_Replica_Tracker(replica_tracker* d_rep, unsigned int node_size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ replica_tracker shared_rep[TPB];
    if(idx<node_size){
        shared_rep[tid]=d_rep[idx];
    }
    __syncthreads();
    if(idx<node_size){
        for(int i=0; i<BLOCKS; i++){
            if(shared_rep[tid].clusters[i]==1){
                shared_rep[tid].num_replicas++;
            }
        }
    }
    __syncthreads();
    if(idx<node_size){
        d_rep[idx]=shared_rep[tid];
    }
}


__global__ void Histogram_1(edge* edgelist, unsigned int* hist_bin, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ unsigned int s_edge_list[TPB];
    __shared__ unsigned int s_hist[BLOCKS];
    if(idx<size){
        s_edge_list[tid]=edgelist[idx].cluster;
        //Copy TPB cluster values over
    }
    __syncthreads();
    if(tid<BLOCKS){
        s_hist[tid]=0;
        //Initialize the histogram
    }
    if(idx<size){
        atomicAdd(s_hist+s_edge_list[tid],1);
        //Increment the histogram based on the cluster value in s_edge_list
    }
    __syncthreads();
    if(tid<BLOCKS){
        hist_bin[gridDim.x*tid+blockIdx.x]=s_hist[tid];
        //Store values for cluster x in block x
    }
    __syncthreads();
    //Now, all the data is stored locally on a blocks/grid by BLOCKS array which we need to reduce
}

__global__ void Kogge_Stone_Hist_Reduct(unsigned int* hist_bin, unsigned int* fin_bin, int size){
    unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ unsigned int clust_val[];
    if(idx<size){ 
        clust_val[tid]=hist_bin[idx];
    }
    else{
        clust_val[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=clust_val[tid]+clust_val[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            clust_val[tid]=temp;
        }
    }
    __syncthreads();
    if(tid==blockDim.x-1){
        fin_bin[blockIdx.x]=clust_val[tid];
    }
    __syncthreads();
}

__global__ void Hist_Prefix_Sum(unsigned int* fin_bin, unsigned int* fin_bin_2){
    unsigned int tid = threadIdx.x+blockDim.x*blockIdx.x;
    __shared__ unsigned int local[BLOCKS];
    if(tid<BLOCKS && tid!=0){
        local[tid]=fin_bin[tid-1];
    }
    else{
        local[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=local[tid]+local[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            local[tid]=temp;
        }
    }
    if(tid<BLOCKS){
        fin_bin_2[tid]=local[tid];
    }
}




__global__ void gen_backward_mask(unsigned int* global_list, unsigned int* ptr_table, unsigned int* ctr_table, unsigned int* start_mask, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ unsigned int start[];
    extern __shared__ unsigned int start_back_mask[];
    if(idx<size){
        //Check that the ctr table is doing what we want
        for(int i=tid; i<2*ctr_table[blockIdx.x];i+=blockDim.x){
            start[i]=global_list[2*ptr_table[blockIdx.x]+i];
        }
    }
    __syncthreads();
    if(idx<size){
        /*Now, we need to generate the hash values*/
        /*We will utilize run length encoding to find the unique values*/
        for(int i = tid; i<2*ctr_table[blockIdx.x];i+=blockDim.x){
            if(i==0){
                start_back_mask[i]=1;
            }
            else{
                if(start[i]!=start[i-1]){
                    start_back_mask[i]=1;
                }
                else{
                    start_back_mask[i]=0;
                }
            }
        }
    }
    __syncthreads();
    /*We have the mask, now, we need to commit to global memory for next kernel*/
    for(int i=tid; i<2*ctr_table[blockIdx.x];i+=blockDim.x){
        start_mask[2*ptr_table[blockIdx.x]+i]=start_back_mask[i];
    }
}


__global__ void scan_mask(unsigned int* start_mask, unsigned* compct_start, unsigned int* ptr_table, unsigned int* ctr_table, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    //We need to use global memory if we intend to use dynamic parallelism, so we need to copy the data over
    /*Now, we can execute the exclusive scan- issue will be that this will be larger than the size of a thread block
    Can we use dynamic parallelism in order to compute partial sums to then acquire a final sum?*/
    int num_of_blocks = (2*ctr_table[blockIdx.x]/blockDim.x)+1;
    if(tid<num_of_blocks){
        int dym_size=(tid==num_of_blocks-1)?(2*ctr_table[blockIdx.x]-tid*blockDim.x):(blockDim.x);
        Prefix_Scan_Cmpt<<<1,blockDim.x,dym_size*sizeof(unsigned int)>>>(start_mask+2*ptr_table[blockIdx.x]+tid*blockDim.x, compct_start+2*ptr_table[blockIdx.x]+tid*blockDim.x,dym_size);
    }
    __syncthreads();
    extern __shared__ unsigned int end_vals[];
    /*Now, we have partial sums, we need to find the final value of each accumulated sum*/
    /*How do we do this..... SUBLIME!*/
    if(tid<num_of_blocks){
        int loc = (tid==num_of_blocks-1)? (2*ptr_table[blockIdx.x+1]-1):(2*ptr_table[blockIdx.x]+(tid+1)*blockDim.x-1);
        end_vals[tid]=compct_start[loc];
    }
    __syncthreads();
    if(tid<num_of_blocks){
        int dym_size=num_of_blocks;
        //Check this
        Prefix_Scan_Cmpt<<<1,num_of_blocks, dym_size*sizeof(unsigned int)>>>(end_vals, end_vals,dym_size);
    }
    __syncthreads();
    if(tid<num_of_blocks){
        int dym_size=(tid==num_of_blocks-1)?(2*ctr_table[blockIdx.x]-tid*blockDim.x):(blockDim.x);
        final_scan_commit_scan<<<1,blockDim.x>>>(compct_start+2*ptr_table[blockIdx.x]+tid*blockDim.x,end_vals, tid, dym_size);
    }
    
}


__global__ void Prefix_Scan_Cmpt(unsigned int* mask, unsigned int* cmpt, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ unsigned int local_cmpt[];
    if(tid<size){
        local_cmpt[tid]=mask[tid];
    }
    __syncthreads();
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=local_cmpt[tid]+local_cmpt[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            local_cmpt[tid]=temp;
        }
    }
    if(idx<size){
        cmpt[idx]=local_cmpt[tid];
    }
    __syncthreads();
}


/*CHECK THIS*/

__global__ void Scanned_To_Compact(unsigned int* cmpt, unsigned int* scanned, unsigned int* new_size, unsigned int* ptr_table, unsigned int* ctr_table, unsigned int size){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    if(idx<size){
        for(int i = tid; i<2*ctr_table[blockIdx.x];i+=blockDim.x){
            if(i==0){
                cmpt[2*ptr_table[blockIdx.x]+i]=0;
            }
            if(i==2*ctr_table[blockIdx.x]-1){
                cmpt[2*ptr_table[blockIdx.x]+scanned[2*ptr_table[blockIdx.x]+i]]=i+1;
                *(new_size+blockIdx.x)=scanned[2*ptr_table[blockIdx.x]+i];
            }
            else if(scanned[2*ptr_table[blockIdx.x]+i]!=scanned[2*ptr_table[blockIdx.x]+i-1]){
                cmpt[scanned[2*ptr_table[blockIdx.x]+i]-1]=i;
            }
        }
    }
}

__global__ void Final_Compression(unsigned int* cmpt, unsigned int* new_size, unsigned int* in, unsigned int* new_idx, unsigned int* out, unsigned int* ptr){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    for(int i = tid; i<(*new_size+blockIdx.x);i+=blockDim.x){
        out[i+2*ptr[blockIdx.x]]=in[cmpt[i+2*ptr[blockIdx.x]]];
        new_idx[i+2*ptr[blockIdx.x]]=cmpt[i+1+2*ptr[blockIdx.x]]-cmpt[i+2*ptr[blockIdx.x]];
    } 
}


__global__ void acc_accum(unsigned int* approx, unsigned int* pagerank, unsigned int* table, unsigned int k){
    unsigned int idx=threadIdx.x + (blockIdx.x*blockDim.x);
    unsigned int tid = threadIdx.x;
    __shared__ unsigned int local_table[TPB];
    if(idx<k){
        if(approx[idx]==pagerank[idx]){
            local_table[tid]=1;
        }
        else{
            local_table[tid]=0;
        }
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=local_table[tid]+local_table[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            local_table[tid]=temp;
        }
    }
    if(tid==(blockDim.x-1)){
        local_table[tid]=table[idx];
    }
}

__global__ void fin_acc(unsigned int* table, unsigned int k, float* acc){
    unsigned int tid = threadIdx.x;
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=table[tid]+table[tid-stride];    
        //Copy TPB cluster values over
    }
    __syncthreads();
        if(tid>=stride){
            table[tid]=temp;
        }
    }
    if(tid==(blockDim.x-1)){
        *acc=float(table[tid])/float(k);
    }
}

__global__ void Find_Max_Cluster(unsigned int* ctr_table, unsigned int* max_val){
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int tid = threadIdx.x;
    __shared__ unsigned int local_max[BLOCKS];
    if(idx<BLOCKS){
        local_max[tid]=ctr_table[idx];
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride && (tid + 1)%(stride*2)==0){
            temp=(local_max[tid]>local_max[tid-stride])?local_max[tid]:local_max[tid-stride];
        }
        __syncthreads();
        if(tid>=stride && (tid + 1)%(stride*2)==0){
            local_max[tid]=temp;
        }
    }
    __syncthreads();
    if(tid==blockDim.x-1){
        *max_val=local_max[tid-1];
    }
}

__global__ void unq_exclusive_scan(unsigned int* len, unsigned int* unq_ptr){
    unsigned int tid=threadIdx.x;
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    __shared__ unsigned int local_ptr_val[BLOCKS];
    if(idx<BLOCKS && idx!=0){
        local_ptr_val[tid]=len[idx-1];
    }
    else{
        local_ptr_val[tid]=0;
    }
    for(unsigned int stride = 1; stride<blockDim.x;stride*=2){
        __syncthreads();
        unsigned int temp;
        if(tid>=stride){
            temp=local_ptr_val[tid]+local_ptr_val[tid-stride];
        }
        __syncthreads();
        if(tid>=stride){
            local_ptr_val[tid]=temp;
        }
    }
    if(idx<BLOCKS){
        unq_ptr[idx]=local_ptr_val[tid];
    }
    __syncthreads();
}


__global__ void Find_Length_of_Unique(unsigned int* start_len, unsigned int* end_len, unsigned int* vector_length){
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    __shared__ unsigned int local_size[BLOCKS];
    if(idx<BLOCKS){
        local_size[idx]=start_len[idx]+end_len[idx];
    }

}


__global__ void Naive_Merge_Sort(unsigned int* start, unsigned int* end, unsigned int* ptr_table, unsigned int* ctr_table, unsigned int* unq){
    //Get the index values for each thread
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    unsigned int tid = threadIdx.x;
    //Find the local start and end values
    unsigned int* local_start=start+ptr_table[blockIdx.x];
    unsigned int* local_end=end+ptr_table[blockIdx.x];
    unsigned int* local_unq=unq+2*ptr_table[blockIdx.x];
    unsigned int elem_per_thread = (ctr_table[blockIdx.x]/blockDim.x)+1;
    unsigned int k_curr = tid*elem_per_thread; //Check that this makes sense
    unsigned int k_next = (tid+1)*elem_per_thread<=ctr_table[blockIdx.x]?(tid+1)*elem_per_thread:ctr_table[blockIdx.x];
    unsigned int i_curr =co_rank(local_start, local_end,ctr_table[blockIdx.x],ctr_table[blockIdx.x],k_curr);
    unsigned int i_next =co_rank(local_start, local_end,ctr_table[blockIdx.x],ctr_table[blockIdx.x],k_next);
    int j_curr = k_curr-i_curr;
    int j_next = k_next-i_next;
    merge_sequential(local_start+i_curr, local_end+j_curr, i_next-i_curr, j_next-j_curr,local_unq+k_curr);
}

__global__ void temp_Copy_Start_End(edge* edge_list, unsigned int* start, unsigned int* end, unsigned int edge_size){
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    if(idx<edge_size){
        start[idx]=edge_list[idx].start;
        end[idx]=edge_list[idx].end;
    }
}

__global__ void Collect_Num_Replicas(replica_tracker* rep, unsigned int* rep_counts, unsigned int size){
    unsigned int idx = threadIdx.x + (blockDim.x*blockIdx.x);
    if(idx<size){
        rep_counts[idx]=rep[idx].num_replicas;
    }
}

__host__ void Org_Vertex_Helper(edge* h_edge, unsigned int* replica_count, replica_tracker* h_tracker, unsigned int* h_deg, unsigned int* h_ctr, unsigned int* h_ptr,unsigned int size, unsigned int node_size){
    //Allocate memory for vertex and cluster info
    edge* d_edge;
    edge* d_edge_2;
    replica_tracker *d_tracker;
    unsigned int* d_table;
    unsigned int* d_table_2;
    unsigned int* d_table_3;

    unsigned int threads_per_block=TPB;
    unsigned int blocks_per_grid= size/threads_per_block+1;
    unsigned int blocks_per_grid_node = node_size/threads_per_block+1;
    cout<<"Num of blocks "<<blocks_per_grid<<endl;
    unsigned int ex_block_pg=(2*blocks_per_grid)/threads_per_block+1;
    cout<<"Second amount of blocks "<< ex_block_pg <<endl;
    cout<<"Allocating d_edge"<<endl;
    if(!HandleCUDAError(hipMalloc((void**) &d_edge, size*sizeof(edge)))){
        cout<<"Unable to allocate memory for vertex data"<<endl;
    }
    cout<<"Copying edge list"<<endl;
    if(!HandleCUDAError(hipMemcpy(d_edge,h_edge,size*sizeof(edge), hipMemcpyHostToDevice))){
        cout<<"Unable to copy cluster data"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**) &d_edge_2, size*sizeof(edge)))){
        cout<<"Unable to allocate memory for vertex data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_edge_2,h_edge,size*sizeof(edge), hipMemcpyHostToDevice))){
        cout<<"Unable to copy cluster data"<<endl;
    }
    cout<<"Done with edge list"<<endl;
    if(!HandleCUDAError(hipMalloc((void**)&d_tracker, node_size*sizeof(replica_tracker)))){
        cout<<"Unable to allocate memory for tracker"<<endl;
    }

    unsigned int* d_degree;
    unsigned int* d_hist;
    unsigned int* dev_fin_hist;
    unsigned int* dev_fin_count;
    unsigned int* max_val;
    unsigned int h_max_val=0;
    // unsigned int* h_hist= new unsigned int [BLOCKS*blocks_per_grid];
    
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
                static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }


    int dev, driverVersion = 0, runtimeVersion = 0;

    for (dev = 0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  Total amount of constant memory:               %zu bytes\n",
            deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %zu bytes\n",
            deviceProp.sharedMemPerBlock);
        printf("  Total shared memory per multiprocessor:        %zu bytes\n",
            deviceProp.sharedMemPerMultiprocessor);
    }

    if(!HandleCUDAError(hipMalloc((void**)&d_hist, BLOCKS*blocks_per_grid*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_hist,0,BLOCKS*blocks_per_grid*sizeof(unsigned int)))){
        cout<<"Unable to set histogram to 0"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&max_val, sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&dev_fin_hist, BLOCKS*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&dev_fin_count, BLOCKS*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for histogram"<<endl;
    }
    if(!HandleCUDAError(hipMalloc((void**)&d_degree, node_size*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for degree"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(d_degree,h_deg,node_size*sizeof(unsigned int), hipMemcpyHostToDevice))){
        cout<<"Unable to copy degree data"<<endl;
    }
    double r = ( ((double)rand())/(RAND_MAX));
    cout<<"The random number is "<<r<<endl;
    cout<<"Starting random edge placement"<<endl;
    Degree_Based_Placement<<<blocks_per_grid,threads_per_block>>>(d_edge,d_degree,r,d_tracker,size);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Rand_Edge Place"<<endl;
    }
    // Random_Edge_Placement<<<blocks_per_grid,threads_per_block>>>(d_edge,r,size);
    // if(!HandleCUDAError(hipDeviceSynchronize())){
    //         cout<<"Unable to synchronize with host with Rand_Edge Place"<<endl;
    // }

    // if(!HandleCUDAError(hipMalloc((void**) &d_table,(2*blocks_per_grid)*sizeof(unsigned int)))){
    //     cout<<"Unable to allocate memory for the table data"<<endl;
    // }
    // if(!HandleCUDAError(hipMemset(d_table,0,(2*blocks_per_grid)*sizeof(unsigned int)))){
    //     cout<<"Unable to set table to 0"<<endl;
    // }

    // if(!HandleCUDAError(hipMalloc((void**) &d_table_2,(2*blocks_per_grid)*sizeof(unsigned int)))){
    //     cout<<"Unable to allocate memory for the table data"<<endl;
    // }
    // if(!HandleCUDAError(hipMemset(d_table_2,0,(2*blocks_per_grid)*sizeof(unsigned int)))){
    //     cout<<"Unable to set table to 0"<<endl;
    // }

    // if(!HandleCUDAError(hipMalloc((void**) &d_table_3,(ex_block_pg)*sizeof(unsigned int)))){
    //     cout<<"Unable to allocate memory for the table data"<<endl;
    // }
    // if(!HandleCUDAError(hipMemset(d_table_3,0,(ex_block_pg)*sizeof(unsigned int)))){
    //     cout<<"Unable to set table to 0"<<endl;
    // }
    // if(ex_block_pg>0){
    //     for(unsigned int i=0; i<=(unsigned int)log2(double(BLOCKS));i++){
    //         cout<<"Iteration "<<i<<endl;
    //         Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
    //         }
    //         bit_exclusive_scan<<<ex_block_pg,threads_per_block>>>(d_table,d_table_2,d_table_3,2*blocks_per_grid);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host exclusive scan"<<endl;
    //         }
    //         fin_exclusive_scan<<<1,ex_block_pg,sizeof(int)*ex_block_pg>>>(d_table_3,ex_block_pg);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host for final exclusive scan"<<endl;
    //         }
    //         final_scan_commit<<<ex_block_pg,threads_per_block>>>(d_table_2,d_table_3,2*blocks_per_grid);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host for final exclusive scan commit"<<endl;
    //         }
    //         Swap<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2,d_table, d_table_2,size, i);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host swap"<<endl;
    //         }
    //         copy_edge_list<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2,size);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host swap"<<endl;
    //         }
    //     }
    // }
    // else{
    //     for(unsigned int i=0; i<(unsigned int)log2(double(BLOCKS));i++){
    //         Sort_Cluster<<<blocks_per_grid,threads_per_block>>>(d_edge,d_table,size,i);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host with Sort Cluster"<<endl;
    //         }
    //         bit_exclusive_scan<<<ex_block_pg,threads_per_block>>>(d_table,d_table_2,d_table_3,2*blocks_per_grid);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host exclusive scan"<<endl;
    //         }
    //         Swap<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2, d_table,d_table_2,size, i);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host swap"<<endl;
    //         }
    //         copy_edge_list<<<blocks_per_grid,threads_per_block>>>(d_edge,d_edge_2,size);
    //         if(!HandleCUDAError(hipDeviceSynchronize())){
    //             cout<<"Unable to synchronize with host swap"<<endl;
    //         }
    //     }
    // }
    // cout<<"Done with sorting"<<endl;
    // HandleCUDAError(hipFree(d_edge_2));
    // HandleCUDAError(hipFree(d_table));
    // HandleCUDAError(hipFree(d_table_2));
    // HandleCUDAError(hipFree(d_table_3));
    hipFuncSetAttribute(reinterpret_cast<const void*>(Finalize_Replica_Tracker), hipFuncAttributeMaxDynamicSharedMemorySize, 102400);
    Finalize_Replica_Tracker<<<blocks_per_grid_node,threads_per_block>>>(d_tracker,node_size);
    if(!HandleCUDAError(hipDeviceSynchronize())){
            cout<<"Unable to synchronize with host with Finalize_Replica_Tracker"<<endl;
    }
    unsigned int* d_replica_counts; //Get the number of replicas for graphs
    if(!HandleCUDAError(hipMalloc((void**)&d_replica_counts, node_size*sizeof(unsigned int)))){
        cout<<"Unable to allocate memory for replica counts"<<endl;
    }
    if(!HandleCUDAError(hipMemset(d_replica_counts,0,node_size*sizeof(unsigned int)))){
        cout<<"Unable to set replica counts to 0"<<endl;
    }
    Collect_Num_Replicas<<<blocks_per_grid_node,threads_per_block>>>(d_tracker,d_replica_counts,node_size);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Unable to synchronize with host with Collect_Num_Replicas"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_tracker,d_tracker,node_size*sizeof(replica_tracker), hipMemcpyDeviceToHost))){
        cout<<"Unable to copy tracker data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(replica_count,d_replica_counts,node_size*sizeof(unsigned int), hipMemcpyDeviceToHost))){
        cout<<"Unable to copy replica counts"<<endl;
    }
    Histogram_1<<<blocks_per_grid,threads_per_block>>>(d_edge,d_hist,size); 
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Unable to synchronize with host with Hist_1"<<endl;
    }
    Kogge_Stone_Hist_Reduct<<<BLOCKS,blocks_per_grid, blocks_per_grid*sizeof(unsigned int)>>>(d_hist,dev_fin_hist,BLOCKS*blocks_per_grid);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Unable to synchronize with host for reduce"<<endl;
    }
    Hist_Prefix_Sum<<<1,BLOCKS>>>(dev_fin_hist, dev_fin_count);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Unable to synchronize with host for reduce"<<endl;
    }
    Find_Max_Cluster<<<1,BLOCKS>>>(dev_fin_hist, max_val);
    if(!HandleCUDAError(hipDeviceSynchronize())){
        cout<<"Unable to synchronize with host for finding the max num of clusters"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(&h_max_val,max_val,sizeof(unsigned int), hipMemcpyDeviceToHost))){
        cout<<"Unable to copy max val"<<endl;
    }
    unsigned int* h_hist_bin;
    h_hist_bin = new unsigned int [BLOCKS*blocks_per_grid];
    if(!HandleCUDAError(hipMemcpy(h_hist_bin,d_hist,BLOCKS*blocks_per_grid*sizeof(unsigned int), hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back ctr"<<endl;
    }
    HandleCUDAError(hipFree(max_val));
    HandleCUDAError(hipFree(d_hist));

    if(!HandleCUDAError(hipMemcpy(h_edge,d_edge,size*sizeof(edge),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back edge data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_ctr,dev_fin_hist,BLOCKS*sizeof(unsigned int),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back ctr data"<<endl;
    }
    if(!HandleCUDAError(hipMemcpy(h_ptr,dev_fin_count,BLOCKS*sizeof(unsigned int),hipMemcpyDeviceToHost))){
        cout<<"Unable to copy back ptr data"<<endl;
    }
    HandleCUDAError(hipFree(d_edge));
    HandleCUDAError(hipFree(d_degree));
    HandleCUDAError(hipFree(d_tracker));
    HandleCUDAError(hipFree(d_replica_counts));
    HandleCUDAError(hipFree(dev_fin_hist));
    HandleCUDAError(hipFree(dev_fin_count));
    HandleCUDAError(hipDeviceReset());   
}


__device__ __host__ void merge_sequential(unsigned int* start, unsigned int* end, int m, int n, unsigned int* unq){
    int i=0;
    int j=0;
    int k=0;
    while(i<m && j<n){
        if(start[i]<=end[j]){
            unq[k]=start[i];
            i++;
            k++;
        }
        else{
            unq[k]=end[j];
            j++;
            k++;
        }
    }
    while(i<m){
        unq[k]=start[i];
        i++;
        k++;
    }
    while(j<n){
        unq[k]=end[j];
        j++;
        k++;
    }
    
}


__device__ unsigned int co_rank(unsigned int* start, unsigned int* end, int m, int n, int k){
    int i = k<m ? k:m;
    int j = k-i;
    int i_low = 0>(k-n) ? 0: k-n;
    int j_low = 0>(k-m) ? 0: k-m;
    int dlt;
    bool done = false;
    while(!done){
        if(i>0 && j<n && start[i-1]>end[j]){
            dlt = (i-i_low+1)/2;
            j_low=j;
            j=j+dlt;
            i=i-dlt;
        }
        else if(j>0 && i<m && end[j-1]>=start[i]){
            dlt = (j-j_low+1)/2;
            i_low=i;
            i=i+dlt;
            j=j-dlt;
        }
        else{
            done=true;
        }
    }
    return i;
}