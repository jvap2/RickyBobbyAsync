﻿#include "../include/GPUErrors.h"

bool HandleCUDAError(hipError_t t)
{
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(hipGetLastError())<<endl;//This will get the string of the error for blocking error
		cout<<t<<endl;
		return false;
	}
	return true;
}
//We can have runtime errors on the GPU, which is what the function below is used for
bool GetCUDARunTimeError()
{
	hipError_t t = hipGetLastError();
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(t) << endl;
		return false;
	}
	return true;
}

bool HandleCUSparseError(hipsparseStatus_t t){
	if (t != HIPSPARSE_STATUS_SUCCESS)
	{
		cout << "CUSPARSE ERROR: " << t << endl;
		cout<< hipsparseGetErrorString(t)<<endl;
		return false;
	}
	return true;
}

bool HandleCUBLASError(hipblasStatus_t t){
	if (t != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS ERROR: " << t << endl;
		cout<< cublasGetErrorString(t)<<endl;
		return false;
	}
	return true;
}

