﻿#include "../include/GPUErrors.h"

bool HandleCUDAError(hipError_t t)
{
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(hipGetLastError())<<endl;//This will get the string of the error for blocking error
		cout<<t<<endl;
		return false;
	}
	return true;
}
//We can have runtime errors on the GPU, which is what the function below is used for
bool GetCUDARunTimeError()
{
	hipError_t t = hipGetLastError();
	if (t != hipSuccess)
	{
		cout << hipGetErrorString(t) << endl;
		return false;
	}
	return true;
}

